#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
  Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus,
  Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl

  This program includes Numerical Recipes (NR) based routines whose
  copyright is held by the NR authors. If NR routines are included,
  you are required to comply with the licensing set forth there.

	Part of the program also relies on an an ANSI C library for multi-stream
	random number generation from the related Prentice-Hall textbook
	Discrete-Event Simulation: A First Course by Steve Park and Larry Leemis,
  for more information please contact leemis@math.wm.edu

  Additionally, this program uses some NVIDIA routines whose copyright is held
  by NVIDIA end user license agreement (EULA).

  For the original parts of this code, the following license applies:

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program. If not, see <http://www.gnu.org/licenses/>.
* -------------------------------------------------------------------------
*/

#include "frprmn.cuh"
#include "directioncosines.cuh"
#include <time.h>

long M, N, numVisibilities;
int iter=0;

hipfftHandle plan1GPU;

hipfftComplex *device_V, *device_Inu;

float2 *device_dphi, *device_2I;
float *device_dS, *device_dS_alpha, *device_chi2, *device_dchi2, *device_S, *device_S_alpha, DELTAX, DELTAY, deltau, deltav, beam_noise, beam_bmaj, nu_0, *device_noise_image, *device_weight_image;
float beam_bmin, b_noise_aux, noise_cut, MINPIX, minpix, lambda, ftol, random_probability;
float noise_jypix, fg_scale, final_chi2, final_H, beam_fwhm, beam_freq, beam_cutoff, alpha_start, eta, epsilon, threshold;

dim3 threadsPerBlockNN;
dim3 numBlocksNN;

int threadsVectorReduceNN, blocksVectorReduceNN, crpix1, crpix2, nopositivity = 0, verbose_flag = 0, clip_flag = 0, apply_noise = 0, print_images = 0, print_errors = 0, gridding, it_maximum, status_mod_in;
int num_gpus, multigpu, firstgpu, selected, t_telescope, reg_term;
char *output, *mempath, *out_image;

double ra, dec;

freqData data;

fitsfile *mod_in;

Field *fields;

VariablesPerField *vars_per_field;

varsPerGPU *vars_gpu;

inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
  #ifdef _WIN32
      return (bool)(pProp->tccDriver ? true : false);
  #else
      return (bool)(pProp->major >= 2);
  #endif
}

inline bool IsAppBuiltAs64()
{
  #if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
      return 1;
  #else
      return 0;
  #endif
}

__host__ int main(int argc, char **argv) {
  clock_t t;
  double start, end;
	////CHECK FOR AVAILABLE GPUs
	hipGetDeviceCount(&num_gpus);

  printf("gpuvmem Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus, Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl\n");
  printf("This program comes with ABSOLUTELY NO WARRANTY; for details use option -w\n");
  printf("This is free software, and you are welcome to redistribute it under certain conditions; use option -c for details.\n\n\n");


	if(num_gpus < 1){
		printf("No CUDA capable devices were detected\n");
    return 1;
	}

	if (!IsAppBuiltAs64()){
        printf("%s is only supported with on 64-bit OSs and the application must be built as a 64-bit target. Test is being waived.\n", argv[0]);
        exit(EXIT_SUCCESS);
  }


	Vars variables = getOptions(argc, argv);
	char *msinput = variables.input;
	char *msoutput = variables.output;
  char *inputdat = variables.inputdat;
	char *modinput = variables.modin;
  out_image = variables.output_image;
  selected = variables.select;
  mempath = variables.path;
  it_maximum = variables.it_max;
  int total_visibilities = 0;
  b_noise_aux = variables.noise;
  lambda = variables.lambda;
  minpix = variables.minpix;
  noise_cut = variables.noise_cut;
  random_probability = variables.randoms;
  reg_term = variables.reg_term;
  nu_0 = variables.nu_0;
  alpha_start = variables.alpha_start;
  eta = variables.eta;
  epsilon = variables.epsilon;
  threshold = variables.threshold * 5.0;
  gridding = variables.gridding;

  multigpu = 0;
  firstgpu = -1;

  struct stat st = {0};
  if(print_images)
    if(stat(mempath, &st) == -1) mkdir(mempath,0700);

  if(verbose_flag){
  	printf("Number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("Number of CUDA devices:\t%d\n", num_gpus);


  	for(int i = 0; i < num_gpus; i++){
    	hipDeviceProp_t dprop;
      hipGetDeviceProperties(&dprop, i);

      printf("> GPU%d = \"%15s\" %s capable of Peer-to-Peer (P2P)\n", i, dprop.name, (IsGPUCapableP2P(&dprop) ? "IS " : "NOT"));

      //printf("   %d: %s\n", i, dprop.name);
    }
    printf("---------------------------\n");
  }

  if(selected > num_gpus || selected < 0){
    printf("ERROR. THE SELECTED GPU DOESN'T EXIST\n");
    exit(-1);
  }

  readInputDat(inputdat);
  init_beam(t_telescope);
  if(verbose_flag){
	   printf("Counting data for memory allocation\n");
  }

  canvasVariables canvas_vars = readCanvas(modinput, mod_in, b_noise_aux, status_mod_in, verbose_flag);

  M = canvas_vars.M;
  N = canvas_vars.N;
  DELTAX = canvas_vars.DELTAX;
  DELTAY = canvas_vars.DELTAY;
  ra = canvas_vars.ra;
  dec = canvas_vars.dec;
  crpix1 = canvas_vars.crpix1;
  crpix2 = canvas_vars.crpix2;
  beam_bmaj = canvas_vars.beam_bmaj;
  beam_bmin = canvas_vars.beam_bmin;
  beam_noise = canvas_vars.beam_noise;

  data = countVisibilities(msinput, fields);

  vars_per_field = (VariablesPerField*)malloc(data.nfields*sizeof(VariablesPerField));

  if(verbose_flag){
     printf("Number of fields = %d\n", data.nfields);
	   printf("Number of frequencies = %d\n", data.total_frequencies);
  }

  if(strcmp(variables.multigpu, "NULL")!=0){
    //Counts number of gpus to use
    char *pt;
    pt = strtok(variables.multigpu,",");

    while(pt!=NULL){
      if(multigpu==0){
        firstgpu = atoi(pt);
      }
      multigpu++;
      pt = strtok (NULL, ",");
    }
  }else{
    multigpu = 0;
  }

  if(multigpu < 0 || multigpu > num_gpus){
    printf("ERROR. NUMBER OF GPUS CANNOT BE NEGATIVE OR GREATER THAN THE NUMBER OF GPUS\n");
    exit(-1);
  }else{
    if(multigpu == 0){
      num_gpus = 1;
    }else{
      if(data.total_frequencies == 1){
        printf("ONLY ONE FREQUENCY. CHANGING NUMBER OF GPUS TO 1\n");
				num_gpus = 1;
      }else{
        num_gpus = multigpu;
        omp_set_num_threads(num_gpus);
      }
    }
  }

 //printf("number of FINAL host CPUs:\t%d\n", omp_get_num_procs());
 if(verbose_flag){
   printf("Number of CUDA devices and threads: \t%d\n", num_gpus);
 }

 //Check peer access if there is more than 1 GPU
  if(num_gpus > 1){
	  for(int i=firstgpu + 1; i< firstgpu + num_gpus; i++){
			hipDeviceProp_t dprop0, dpropX;
			hipGetDeviceProperties(&dprop0, firstgpu);
			hipGetDeviceProperties(&dpropX, i);
			int canAccessPeer0_x, canAccessPeerx_0;
			hipDeviceCanAccessPeer(&canAccessPeer0_x, firstgpu, i);
			hipDeviceCanAccessPeer(&canAccessPeerx_0 , i, firstgpu);
      if(verbose_flag){
  			printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", dprop0.name, firstgpu, dpropX.name, i, canAccessPeer0_x ? "Yes" : "No");
      	printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", dpropX.name, i, dprop0.name, firstgpu, canAccessPeerx_0 ? "Yes" : "No");
      }
			if(canAccessPeer0_x == 0 || canAccessPeerx_0 == 0){
				printf("Two or more SM 2.0 class GPUs are required for %s to run.\n", argv[0]);
        printf("Support for UVA requires a GPU with SM 2.0 capabilities.\n");
        printf("Peer to Peer access is not available between GPU%d <-> GPU%d, waiving test.\n", 0, i);
        exit(EXIT_SUCCESS);
			}else{
				hipSetDevice(firstgpu);
        if(verbose_flag){
          printf("Granting access from %d to %d...\n",firstgpu, i);
        }
				hipDeviceEnablePeerAccess(i,0);
				hipSetDevice(i);
        if(verbose_flag){
          printf("Granting access from %d to %d...\n", i, firstgpu);
        }
				hipDeviceEnablePeerAccess(firstgpu,0);
        if(verbose_flag){
				      printf("Checking GPU %d and GPU %d for UVA capabilities...\n", firstgpu, i);
        }
				const bool has_uva = (dprop0.unifiedAddressing && dpropX.unifiedAddressing);
        if(verbose_flag){
  				printf("> %s (GPU%d) supports UVA: %s\n", dprop0.name, firstgpu, (dprop0.unifiedAddressing ? "Yes" : "No"));
      		printf("> %s (GPU%d) supports UVA: %s\n", dpropX.name, i, (dpropX.unifiedAddressing ? "Yes" : "No"));
        }
				if (has_uva){
          if(verbose_flag){
        	   printf("Both GPUs can support UVA, enabling...\n");
          }
    		}
    		else{
        	printf("At least one of the two GPUs does NOT support UVA, waiving test.\n");
        	exit(EXIT_SUCCESS);
    		}
			}
	 	}
    vars_gpu = (varsPerGPU*)malloc(num_gpus*sizeof(varsPerGPU));
  }

  for(int f=0; f<data.nfields; f++){
  	fields[f].visibilities = (Vis*)malloc(data.total_frequencies*sizeof(Vis));
    fields[f].gridded_visibilities = (Vis*)malloc(data.total_frequencies*sizeof(Vis));
  	fields[f].device_visibilities = (Vis*)malloc(data.total_frequencies*sizeof(Vis));
  }

  //ALLOCATE MEMORY AND GET TOTAL NUMBER OF VISIBILITIES
  for(int f=0; f<data.nfields; f++){
  	for(int i=0; i < data.total_frequencies; i++){
  		fields[f].visibilities[i].stokes = (int*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(int));
  		fields[f].visibilities[i].u = (float*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(float));
  		fields[f].visibilities[i].v = (float*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(float));
  		fields[f].visibilities[i].weight = (float*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(float));
  		fields[f].visibilities[i].Vo = (hipfftComplex*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(hipfftComplex));
      if(gridding){
    		fields[f].gridded_visibilities[i].u = (float*)malloc(M*N*sizeof(float));
    		fields[f].gridded_visibilities[i].v = (float*)malloc(M*N*sizeof(float));
    		fields[f].gridded_visibilities[i].weight = (float*)malloc(M*N*sizeof(float));
    		fields[f].gridded_visibilities[i].Vo = (hipfftComplex*)malloc(M*N*sizeof(hipfftComplex));

        memset(fields[f].gridded_visibilities[i].u, 0, M*N*sizeof(float));
        memset(fields[f].gridded_visibilities[i].v, 0, M*N*sizeof(float));
        memset(fields[f].gridded_visibilities[i].weight, 0, M*N*sizeof(float));
        memset(fields[f].gridded_visibilities[i].Vo, 0, M*N*sizeof(hipfftComplex));
      }else{
          fields[f].visibilities[i].Vm = (hipfftComplex*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(hipfftComplex));
      }
      fields[f].visibilities[i].Vm = (hipfftComplex*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(hipfftComplex));
  	}
  }


  if(verbose_flag){
	   printf("Reading visibilities and FITS input files...\n");
  }

  if(apply_noise && random_probability < 1.0){
    readMCNoiseSubsampledMS(msinput, fields, data, random_probability);
  }else if(random_probability < 1.0){
    readSubsampledMS(msinput, fields, data, random_probability);
  }else if(apply_noise){
    readMSMCNoise(msinput, fields, data);
  }else{
     readMS(msinput, fields, data);
  }

  float deltax = RPDEG*DELTAX; //radians
  float deltay = RPDEG*DELTAY; //radians
  deltau = 1.0 / (M * deltax);
  deltav = 1.0 / (N * deltay);
  if(gridding){
    omp_set_num_threads(gridding);
    do_gridding(fields, &data, deltau, deltav, M, N, &total_visibilities);
    omp_set_num_threads(num_gpus);
  }

   float sum_weights = calculateNoise(fields, data, &total_visibilities, variables.blockSizeV);
   if(verbose_flag){
     printf("MS File Successfully Read\n");
     if(beam_noise == -1){
       printf("Beam noise wasn't provided by the user... Calculating...\n");
     }
 }

	if(num_gpus == 1){
    hipSetDevice(selected);
    for(int f=0; f<data.nfields; f++){
  		for(int i=0; i<data.total_frequencies; i++){
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].u, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].v, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
        gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vm, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  		}
    }
	}else{
    for(int f=0; f<data.nfields; f++){
  		for(int i=0; i<data.total_frequencies; i++){
  			hipSetDevice((i%num_gpus) + firstgpu);
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].u, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].v, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
        gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vm, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  		}
    }
	}


	if(num_gpus == 1){
    hipSetDevice(selected);
    gpuErrchk(hipMalloc((void**)&device_dchi2, sizeof(float)*M*N));
    gpuErrchk(hipMemset(device_dchi2, 0, sizeof(float)*M*N));

    gpuErrchk(hipMalloc(&device_chi2, sizeof(float)*data.max_number_visibilities_in_channel));
    gpuErrchk(hipMemset(device_chi2, 0, sizeof(float)*data.max_number_visibilities_in_channel));

    for(int f=0; f<data.nfields; f++){
      gpuErrchk(hipMalloc((void**)&vars_per_field[f].atten_image, sizeof(hipfftComplex)*M*N));
      gpuErrchk(hipMemset(vars_per_field[f].atten_image, 0, sizeof(hipfftComplex)*M*N));
  		for(int i=0; i < data.total_frequencies; i++){

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].u, fields[f].visibilities[i].u, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].v, fields[f].visibilities[i].v, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].weight, fields[f].visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

        gpuErrchk(hipMemset(fields[f].device_visibilities[i].Vm, 0, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));

  		}
    }
	}else{

    for(int g=0; g<num_gpus; g++){
      hipSetDevice((g%num_gpus) + firstgpu);
      gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_dchi2, sizeof(float)*M*N));
      gpuErrchk(hipMemset(vars_gpu[g].device_dchi2, 0, sizeof(float)*M*N));

      gpuErrchk(hipMalloc(&vars_gpu[g].device_chi2, sizeof(float)*data.max_number_visibilities_in_channel));
      gpuErrchk(hipMemset(vars_gpu[g].device_chi2, 0, sizeof(float)*data.max_number_visibilities_in_channel));
    }

    for(int f=0; f<data.nfields; f++){
      hipSetDevice(firstgpu);
      gpuErrchk(hipMalloc((void**)&vars_per_field[f].atten_image, sizeof(hipfftComplex)*M*N));
      gpuErrchk(hipMemset(vars_per_field[f].atten_image, 0, sizeof(hipfftComplex)*M*N));
  		for(int i=0; i < data.total_frequencies; i++){
  			hipSetDevice((i%num_gpus) + firstgpu);

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].u, fields[f].visibilities[i].u, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].v, fields[f].visibilities[i].v, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].weight, fields[f].visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

        gpuErrchk(hipMemset(fields[f].device_visibilities[i].Vm, 0, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  		}
    }
	}

  //Declaring block size and number of blocks for Image
  dim3 threads(variables.blockSizeX, variables.blockSizeY);
	dim3 blocks(M/threads.x, N/threads.y);
	threadsPerBlockNN = threads;
	numBlocksNN = blocks;

	noise_jypix = beam_noise / (PI * beam_bmaj * beam_bmin / (4 * log(2) ));
  if(lambda == 0.0){
    MINPIX = 0.0;
  }else{
      MINPIX = minpix;
  }

  /////////////////////////////////////////////////////CALCULATE DIRECTION COSINES/////////////////////////////////////////////////
  double raimage = ra * RPDEG_D;
  double decimage = dec * RPDEG_D;
  if(verbose_flag){
    printf("FITS: Ra: %lf, dec: %lf\n", raimage, decimage);
  }
  for(int f=0; f<data.nfields; f++){
  	double lobs, mobs;

  	direccos(fields[f].obsra, fields[f].obsdec, raimage, decimage, &lobs,  &mobs);

    if(crpix1 != crpix2){
  	  fields[f].global_xobs = (crpix1 - 1.0) - (lobs/deltax) + 1.0;
  	  fields[f].global_yobs = (crpix2 - 1.0) - (mobs/deltay) - 1.0;
    }else{
      fields[f].global_xobs = (crpix1 - 1.0) - (lobs/deltax) - 1.0;
  	  fields[f].global_yobs = (crpix2 - 1.0) - (mobs/deltay) - 1.0;
    }

    if(verbose_flag){
  	   printf("Field %d - Ra: %f, dec: %f , x0: %f, y0: %f\n", f, fields[f].obsra, fields[f].obsdec, fields[f].global_xobs, fields[f].global_yobs);
    }

    if(fields[f].global_xobs < 0 || fields[f].global_xobs > M || fields[f].global_xobs < 0 || fields[f].global_yobs > N) {
      printf("Pointing center (%f,%f) is outside the range of the image\n", fields[f].global_xobs, fields[f].global_xobs);
      goToError();
    }
  }
	////////////////////////////////////////////////////////MAKE STARTING IMAGE////////////////////////////////////////////////////////
	float2 *host_2I = (float2*)malloc(M*N*sizeof(float2));

  int x = M-1;
  int y = N-1;
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
		    host_2I[N*i+j].x = MINPIX;  // I_nu
        host_2I[N*i+j].y = alpha_start;
        x--;
		}
    x=M-1;
    y--;
	}

	////////////////////////////////////////////////CUDA MEMORY ALLOCATION FOR DEVICE///////////////////////////////////////////////////

	if(num_gpus == 1){
    hipSetDevice(selected);
    gpuErrchk(hipMalloc((void**)&device_Inu, sizeof(hipfftComplex)*M*N));
		gpuErrchk(hipMalloc((void**)&device_V, sizeof(hipfftComplex)*M*N));
	}else{
    for(int g = 0; g<num_gpus; g++){
  			hipSetDevice((g%num_gpus) + firstgpu);
        gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_Inu, sizeof(hipfftComplex)*M*N));
  			gpuErrchk(hipMalloc((void**)&vars_gpu[g].device_V, sizeof(hipfftComplex)*M*N));

    }
	}

  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
	   hipSetDevice(firstgpu);
  }

	gpuErrchk(hipMalloc((void**)&device_2I, sizeof(float2)*M*N));
  gpuErrchk(hipMemset(device_2I, 0, sizeof(float2)*M*N));

  gpuErrchk(hipMemcpy2D(device_2I, sizeof(float2), host_2I, sizeof(float2), sizeof(float2), M*N, hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**)&device_noise_image, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_noise_image, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_weight_image, sizeof(hipfftComplex)*M*N));
  gpuErrchk(hipMemset(device_weight_image, 0, sizeof(hipfftComplex)*M*N));

	gpuErrchk(hipMalloc((void**)&device_dphi, sizeof(float2)*M*N));
  gpuErrchk(hipMemset(device_dphi, 0, sizeof(float2)*M*N));

	gpuErrchk(hipMalloc((void**)&device_dS, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_dS, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_dS_alpha, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_dS_alpha, 0, sizeof(float)*M*N));

	gpuErrchk(hipMalloc((void**)&device_S, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_S, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_S_alpha, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_S_alpha, 0, sizeof(float)*M*N));

	if(num_gpus == 1){
    hipSetDevice(selected);
    gpuErrchk(hipMemset(device_Inu, 0, sizeof(hipfftComplex)*M*N));
		gpuErrchk(hipMemset(device_V, 0, sizeof(hipfftComplex)*M*N));
	}else{
    for(int g = 0; g<num_gpus; g++){
  			hipSetDevice((g%num_gpus) + firstgpu);
        gpuErrchk(hipMemset(vars_gpu[g].device_V, 0, sizeof(hipfftComplex)*M*N));
  			gpuErrchk(hipMemset(vars_gpu[g].device_Inu, 0, sizeof(hipfftComplex)*M*N));
    }
	}




	if(num_gpus == 1){
    hipSetDevice(selected);
		if ((hipfftPlan2d(&plan1GPU, N, M, HIPFFT_C2C))!= HIPFFT_SUCCESS) {
			printf("cufft plan error\n");
			return -1;
		}
	}else{
    for(int g = 0; g<num_gpus; g++){
      hipSetDevice((g%num_gpus) + firstgpu);
      if ((hipfftPlan2d(&vars_gpu[g].plan, N, M, HIPFFT_C2C))!= HIPFFT_SUCCESS) {
        printf("cufft plan error\n");
        return -1;
      }
    }
	}

  //Time is taken from first kernel
  t = clock();
  start = omp_get_wtime();
	if(num_gpus == 1){
    hipSetDevice(selected);
    for(int f=0; f < data.nfields; f++){
  		for(int i=0; i<data.total_frequencies; i++){
  			hermitianSymmetry<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].u, fields[f].device_visibilities[i].v, fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].freq, fields[f].numVisibilitiesPerFreq[i]);
  			gpuErrchk(hipDeviceSynchronize());
  		}
    }
	}else{
    for(int f = 0; f < data.nfields; f++){
  		#pragma omp parallel for schedule(static,1)
      for (int i = 0; i < data.total_frequencies; i++)
  		{
  			unsigned int j = omp_get_thread_num();
  			//unsigned int num_cpu_threads = omp_get_num_threads();
  			// set and check the CUDA device for this CPU thread
  			int gpu_id = -1;
  			hipSetDevice((i%num_gpus) + firstgpu);   // "% num_gpus" allows more CPU threads than GPU devices
  			hipGetDevice(&gpu_id);
  			hermitianSymmetry<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].u, fields[f].device_visibilities[i].v, fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].freq, fields[f].numVisibilitiesPerFreq[i]);
  			gpuErrchk(hipDeviceSynchronize());
  		}

  	}
  }

	if(num_gpus == 1){
    hipSetDevice(selected);
    for(int f=0; f<data.nfields; f++){
  		for(int i=0; i<data.total_frequencies; i++){
        if(fields[f].numVisibilitiesPerFreq[i] > 0){
          total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(vars_per_field[f].atten_image, beam_fwhm, beam_freq, beam_cutoff, fields[f].visibilities[i].freq, fields[f].global_xobs, fields[f].global_yobs, DELTAX, DELTAY, N);
    			gpuErrchk(hipDeviceSynchronize());
        }
  		}
    }
	}else{
    for(int f=0; f<data.nfields; f++){
      #pragma omp parallel for schedule(static,1)
      for (int i = 0; i < data.total_frequencies; i++)
  		{
        unsigned int j = omp_get_thread_num();
  			//unsigned int num_cpu_threads = omp_get_num_threads();
  			// set and check the CUDA device for this CPU thread
  			int gpu_id = -1;
  			hipSetDevice((i%num_gpus) + firstgpu);   // "% num_gpus" allows more CPU threads than GPU devices
  			hipGetDevice(&gpu_id);
        if(fields[f].numVisibilitiesPerFreq[i] > 0){
    			#pragma omp critical
    			{
    				total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(vars_per_field[f].atten_image, beam_fwhm, beam_freq, beam_cutoff, fields[f].visibilities[i].freq, fields[f].global_xobs, fields[f].global_yobs, DELTAX, DELTAY, N);
    				gpuErrchk(hipDeviceSynchronize());
    			}
        }
  		}
  	}
  }

  for(int f=0; f<data.nfields; f++){
    if(fields[f].valid_frequencies > 0){
      if(num_gpus == 1){
        hipSetDevice(selected);
      	mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(vars_per_field[f].atten_image, fields[f].valid_frequencies, N);
      	gpuErrchk(hipDeviceSynchronize());
    	}else{
        hipSetDevice(firstgpu);
      	mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(vars_per_field[f].atten_image, fields[f].valid_frequencies, N);
      	gpuErrchk(hipDeviceSynchronize());
    	}
      if(print_images)
        fitsOutputFloat(vars_per_field[f].atten_image, mod_in, mempath, f, M, N, 0);
    }
  }

  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
	   hipSetDevice(firstgpu);
  }

  for(int f=0; f<data.nfields; f++){
    weight_image<<<numBlocksNN, threadsPerBlockNN>>>(device_weight_image, vars_per_field[f].atten_image, noise_jypix, N);
    gpuErrchk(hipDeviceSynchronize());
  }
  noise_image<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_weight_image, noise_jypix, N);
  gpuErrchk(hipDeviceSynchronize());
  if(print_images)
    fitsOutputFloat(device_noise_image, mod_in, mempath, 0, M, N, 1);


	float *host_noise_image = (float*)malloc(M*N*sizeof(float));
	gpuErrchk(hipMemcpy2D(host_noise_image, sizeof(float), device_noise_image, sizeof(float), sizeof(float), M*N, hipMemcpyDeviceToHost));
	float noise_min = *std::min_element(host_noise_image,host_noise_image+(M*N));

	fg_scale = noise_min;
	noise_cut = noise_cut * noise_min;
  if(verbose_flag){
	   printf("fg_scale = %e\n", fg_scale);
     printf("noise_jypix = %e\n", noise_jypix);
  }
	free(host_noise_image);
  hipFree(device_weight_image);
  for(int f=0; f<data.nfields; f++){
    hipFree(vars_per_field[f].atten_image);
  }



	//////////////////////////////////////////////////////Fletcher-Reeves Polak-Ribiere Minimization////////////////////////////////////////////////////////////////
	printf("\n\nStarting Fletcher Reeves Polak Ribiere method (Conj. Grad.)\n\n");
	float fret = 0.0;

	frprmn(device_2I, ftol, &fret, chiCuadrado, dchiCuadrado, 0);
  chiCuadrado(device_2I);
  fret = 0.0;
  frprmn(device_2I, ftol, &fret, chiCuadrado, dchiCuadrado, 1);
  chiCuadrado(device_2I);
  fret = 0.0;
  frprmn(device_2I, ftol, &fret, chiCuadrado, dchiCuadrado, 2);
  chiCuadrado(device_2I);
  fret = 0.0;
  frprmn(device_2I, ftol, &fret, chiCuadrado, dchiCuadrado, 3);
  chiCuadrado(device_2I);
  t = clock() - t;
  end = omp_get_wtime();
  printf("Minimization ended successfully\n\n");
  printf("Iterations: %d\n", iter);
  printf("chi2: %f\n", final_chi2);
  printf("0.5*chi2: %f\n", 0.5*final_chi2);
  printf("Total visibilities: %d\n", total_visibilities);
  printf("Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
  printf("Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
  printf("S: %f\n", final_H);
  if(reg_term != 1){
    printf("Normalized S: %f\n", final_H/(M*N));
  }else{
    printf("Normalized S: %f\n", final_H/(M*M*N*N));
  }
  printf("lambda*S: %f\n\n", lambda*final_H);
	double time_taken = ((double)t)/CLOCKS_PER_SEC;
  double wall_time = end-start;
  printf("Total CPU time: %lf\n", time_taken);
  printf("Wall time: %lf\n\n\n", wall_time);

  if(strcmp(variables.ofile,"NULL") != 0){
    FILE *outfile = fopen(variables.ofile, "w");
    if (outfile == NULL)
    {
        printf("Error opening output file!\n");
        goToError();
    }

    fprintf(outfile, "Iterations: %d\n", iter);
    fprintf(outfile, "chi2: %f\n", final_chi2);
    fprintf(outfile, "0.5*chi2: %f\n", 0.5*final_chi2);
    fprintf(outfile, "Total visibilities: %d\n", total_visibilities);
    fprintf(outfile, "Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
    fprintf(outfile, "Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
    fprintf(outfile, "S: %f\n", final_H);
    if(reg_term != 1){
      fprintf(outfile, "Normalized S: %f\n", final_H/(M*N));
    }else{
      fprintf(outfile, "Normalized S: %f\n", final_H/(M*M*N*N));
    }
    fprintf(outfile, "lambda*S: %f\n", lambda*final_H);
    fprintf(outfile, "Wall time: %lf", wall_time);
    fclose(outfile);
  }
	//Pass residuals to host
	printf("Saving final image to disk\n");
	float2toImage(device_2I, mod_in, out_image, mempath, iter, fg_scale, M, N, 0);
	//Saving residuals to disk
  residualsToHost(fields, data, num_gpus, firstgpu);
  printf("Saving residuals to MS...\n");
	writeMS(msinput, msoutput, fields, data, random_probability, verbose_flag);
	printf("Residuals saved.\n");

	//Free device and host memory
	printf("Free device and host memory\n");
	hipfftDestroy(plan1GPU);
  for(int f=0; f<data.nfields; f++){
  	for(int i=0; i<data.total_frequencies; i++){
      if(num_gpus > 1){
  		    hipSetDevice((i%num_gpus) + firstgpu);
      }
  		hipFree(fields[f].device_visibilities[i].u);
  		hipFree(fields[f].device_visibilities[i].v);
  		hipFree(fields[f].device_visibilities[i].weight);
  		hipFree(fields[f].device_visibilities[i].Vo);
      hipFree(fields[f].device_visibilities[i].Vm);

  	}
  }

  if(num_gpus>1){
    for(int g = 0; g<num_gpus; g++){
        hipSetDevice((g%num_gpus) + firstgpu);
        hipfftDestroy(vars_gpu[g].plan);
    }
  }

  for(int f=0; f<data.nfields; f++){
  	for(int i=0; i<data.total_frequencies; i++){
      if(fields[f].numVisibilitiesPerFreq[i] != 0){
    		free(fields[f].visibilities[i].u);
    		free(fields[f].visibilities[i].v);
    		free(fields[f].visibilities[i].weight);
    		free(fields[f].visibilities[i].Vo);
        free(fields[f].visibilities[i].Vm);
      }
  	}
  }

	hipFree(device_2I);
	if(num_gpus == 1){
		hipFree(device_V);
		hipFree(device_Inu);
	}else{
    for(int g = 0; g<num_gpus; g++){
  			hipSetDevice((g%num_gpus) + firstgpu);
  			hipFree(vars_gpu[g].device_V);
  			hipFree(vars_gpu[g].device_Inu);
  		}
	}
  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
    hipSetDevice(firstgpu);
  }

	hipFree(device_noise_image);

	hipFree(device_dphi);
	hipFree(device_dS);
  hipFree(device_dS_alpha);

	hipFree(device_chi2);
	hipFree(device_S);
  hipFree(device_S_alpha);

  //Disabling UVA
  if(num_gpus > 1){
    for(int i=firstgpu+1; i<num_gpus+firstgpu; i++){
          hipSetDevice(firstgpu);
          hipDeviceDisablePeerAccess(i);
          hipSetDevice(i);
          hipDeviceDisablePeerAccess(firstgpu);
    }

    for(int i=0; i<num_gpus; i++ ){
          hipSetDevice((i%num_gpus) + firstgpu);
          hipDeviceReset();
    }
  }
	free(host_2I);
	free(msinput);
	free(msoutput);
	free(modinput);

  closeCanvas(mod_in);

  if (status_mod_in) {
    fits_report_error(stderr, status_mod_in);
    goToError();
  }

	return 0;
}
