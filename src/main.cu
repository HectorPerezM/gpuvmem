#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
  Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus,
  Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl

  This program includes Numerical Recipes (NR) based routines whose
  copyright is held by the NR authors. If NR routines are included,
  you are required to comply with the licensing set forth there.

	Part of the program also relies on an an ANSI C library for multi-stream
	random number generation from the related Prentice-Hall textbook
	Discrete-Event Simulation: A First Course by Steve Park and Larry Leemis,
  for more information please contact leemis@math.wm.edu

  For the original parts of this code, the following license applies:

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program. If not, see <http://www.gnu.org/licenses/>.
* -------------------------------------------------------------------------
*/

#include "frprmn.cuh"
#include "directioncosines.cuh"
#include <time.h>

long M, N, numVisibilities;
int iter=0;

hipfftHandle plan1GPU;

hipfftComplex *device_V, *device_Inu;

float3 *device_dphi, *device_dchi2_total, *device_3I;
float *device_dS, *device_chi2, *device_S, DELTAX, DELTAY, deltau, deltav, beam_noise, beam_bmaj, nu_0, *device_noise_image, *device_weight_image;
float beam_bmin, b_noise_aux, noise_cut, MINPIX, minpix, lambda, ftol, random_probability;
float difmap_noise, fg_scale, final_chi2, final_H, beam_fwhm, beam_freq, beam_cutoff, freqavg;

dim3 threadsPerBlockNN;
dim3 numBlocksNN;

int threadsVectorReduceNN, blocksVectorReduceNN, crpix1, crpix2, nopositivity = 0, nsamples, nfields, nstokes, verbose_flag = 0, clip_flag = 0, it_maximum, status_mod_in;
int num_gpus, multigpu, firstgpu, selected, t_telescope, reg_term;
char *output, *mempath, *out_image;

double ra, dec;

freqData data;
fitsfile *mod_in;

Field *fields;
inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
  #ifdef _WIN32
      return (bool)(pProp->tccDriver ? true : false);
  #else
      return (bool)(pProp->major >= 2);
  #endif
}

inline bool IsAppBuiltAs64()
{
  #if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
      return 1;
  #else
      return 0;
  #endif
}

__host__ int main(int argc, char **argv) {
  clock_t t;
  double start, end;
	////CHECK FOR AVAILABLE GPUs
	hipGetDeviceCount(&num_gpus);

  printf("gpuvmem Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus, Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl\n");
  printf("This program comes with ABSOLUTELY NO WARRANTY; for details use option -w\n");
  printf("This is free software, and you are welcome to redistribute it under certain conditions; use option -c for details.\n\n\n");


	if(num_gpus < 1){
		printf("No CUDA capable devices were detected\n");
    return 1;
	}

	if (!IsAppBuiltAs64()){
        printf("%s is only supported with on 64-bit OSs and the application must be built as a 64-bit target. Test is being waived.\n", argv[0]);
        exit(EXIT_SUCCESS);
  }


	float noise_min = 1E32;

	Vars variables = getOptions(argc, argv);
	char *msinput = variables.input;
	char *msoutput = variables.output;
  char *inputdat = variables.inputdat;
	char *modinput = variables.modin;
  char *Tinput = variables.Tin;
  out_image = variables.output_image;
  selected = variables.select;
  mempath = variables.path;
  it_maximum = variables.it_max;
  int total_visibilities = 0;
  b_noise_aux = variables.noise;
  lambda = variables.lambda;
  minpix = variables.minpix;
  noise_cut = variables.noise_cut;
  random_probability = variables.randoms;
  reg_term = variables.reg_term;
  nu_0 = variables.nu_0;

  multigpu = 0;
  firstgpu = -1;

  struct stat st = {0};

  if(stat(mempath, &st) == -1) mkdir(mempath,0700);

  if(verbose_flag){
  	printf("Number of host CPUs:\t%d\n", omp_get_num_procs());
    printf("Number of CUDA devices:\t%d\n", num_gpus);


  	for(int i = 0; i < num_gpus; i++){
    	hipDeviceProp_t dprop;
      hipGetDeviceProperties(&dprop, i);

      printf("> GPU%d = \"%15s\" %s capable of Peer-to-Peer (P2P)\n", i, dprop.name, (IsGPUCapableP2P(&dprop) ? "IS " : "NOT"));

      //printf("   %d: %s\n", i, dprop.name);
    }
    printf("---------------------------\n");
  }

  if(selected > num_gpus || selected < 0){
    printf("ERROR. THE SELECTED GPU DOESN'T EXIST\n");
    exit(-1);
  }

  readInputDat(inputdat);
  init_beam(t_telescope);
  if(verbose_flag){
	   printf("Counting data for memory allocation\n");
  }
	data = getFreqs(msinput);
  if(verbose_flag){
	   printf("Number of frequencies file = %d\n", data.total_frequencies);
  }

  if(strcmp(variables.multigpu, "NULL")!=0){
    //Counts number of gpus to use
    char *pt;
    pt = strtok(variables.multigpu,",");

    while(pt!=NULL){
      if(multigpu==0){
        firstgpu = atoi(pt);
      }
      multigpu++;
      pt = strtok (NULL, ",");
    }
  }else{
    multigpu = 0;
  }

  if(multigpu < 0 || multigpu > num_gpus){
    printf("ERROR. NUMBER OF GPUS CANNOT BE NEGATIVE OR GREATER THAN THE NUMBER OF GPUS\n");
    exit(-1);
  }else{
    if(multigpu == 0){
      num_gpus = 1;
    }else{
      if(data.total_frequencies == 1){
        printf("ONLY ONE FREQUENCY. CHANGING NUMBER OF GPUS TO 1\n");
				num_gpus = 1;
      }else{
        num_gpus = multigpu;
        omp_set_num_threads(num_gpus);
      }
    }
  }

 //printf("number of FINAL host CPUs:\t%d\n", omp_get_num_procs());
 if(verbose_flag){
   printf("Number of CUDA devices and threads: \t%d\n", num_gpus);
 }

 //Check peer access if there is more than 1 GPU
  if(num_gpus > 1){
	  for(int i=firstgpu + 1; i< firstgpu + num_gpus; i++){
			hipDeviceProp_t dprop0, dpropX;
			hipGetDeviceProperties(&dprop0, firstgpu);
			hipGetDeviceProperties(&dpropX, i);
			int canAccessPeer0_x, canAccessPeerx_0;
			hipDeviceCanAccessPeer(&canAccessPeer0_x, firstgpu, i);
			hipDeviceCanAccessPeer(&canAccessPeerx_0 , i, firstgpu);
      if(verbose_flag){
  			printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", dprop0.name, firstgpu, dpropX.name, i, canAccessPeer0_x ? "Yes" : "No");
      	printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", dpropX.name, i, dprop0.name, firstgpu, canAccessPeerx_0 ? "Yes" : "No");
      }
			if(canAccessPeer0_x == 0 || canAccessPeerx_0 == 0){
				printf("Two or more SM 2.0 class GPUs are required for %s to run.\n", argv[0]);
        printf("Support for UVA requires a GPU with SM 2.0 capabilities.\n");
        printf("Peer to Peer access is not available between GPU%d <-> GPU%d, waiving test.\n", 0, i);
        exit(EXIT_SUCCESS);
			}else{
				hipSetDevice(firstgpu);
        if(verbose_flag){
          printf("Granting access from %d to %d...\n",firstgpu, i);
        }
				hipDeviceEnablePeerAccess(i,0);
				hipSetDevice(i);
        if(verbose_flag){
          printf("Granting access from %d to %d...\n", i, firstgpu);
        }
				hipDeviceEnablePeerAccess(firstgpu,0);
        if(verbose_flag){
				      printf("Checking GPU %d and GPU %d for UVA capabilities...\n", firstgpu, i);
        }
				const bool has_uva = (dprop0.unifiedAddressing && dpropX.unifiedAddressing);
        if(verbose_flag){
  				printf("> %s (GPU%d) supports UVA: %s\n", dprop0.name, firstgpu, (dprop0.unifiedAddressing ? "Yes" : "No"));
      		printf("> %s (GPU%d) supports UVA: %s\n", dpropX.name, i, (dpropX.unifiedAddressing ? "Yes" : "No"));
        }
				if (has_uva){
          if(verbose_flag){
        	   printf("Both GPUs can support UVA, enabling...\n");
          }
    		}
    		else{
        	printf("At least one of the two GPUs does NOT support UVA, waiving test.\n");
        	exit(EXIT_SUCCESS);
    		}
			}
	 	}
  }

  for(int f=0; f<nfields; f++){
  	fields[f].visibilities = (Vis*)malloc(data.total_frequencies*sizeof(Vis));
  	fields[f].device_visibilities = (Vis*)malloc(data.total_frequencies*sizeof(Vis));
  	fields[f].device_vars = (VPF*)malloc(data.total_frequencies*sizeof(VPF));
  }

  //ALLOCATE MEMORY AND GET TOTAL NUMBER OF VISIBILITIES
  for(int f=0; f<nfields; f++){
  	for(int i=0; i < data.total_frequencies; i++){
  		fields[f].visibilities[i].stokes = (int*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(int));
  		fields[f].visibilities[i].u = (float*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(float));
  		fields[f].visibilities[i].v = (float*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(float));
  		fields[f].visibilities[i].weight = (float*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(float));
  		fields[f].visibilities[i].Vo = (hipfftComplex*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(hipfftComplex));
      fields[f].visibilities[i].Vm = (hipfftComplex*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(hipfftComplex));
      total_visibilities += fields[f].numVisibilitiesPerFreq[i];
  	}
  }



  if(verbose_flag){
	   printf("Reading visibilities and FITS input files...\n");
  }
	readMS(msinput, modinput, fields);

  if(verbose_flag){
    printf("MS File Successfully Read\n");
    if(beam_noise == -1){
      printf("Beam noise wasn't provided by the user... Calculating...\n");
    }
    printf("Calculating weights sum\n");
  }

  freqavg = 1.37995e+11;

  //Declaring block size and number of blocks for visibilities
  float sum_inverse_weight = 0.0;
  float sum_weights = 0.0;
  for(int f=0; f<nfields; f++){
  	for(int i=0; i< data.total_frequencies; i++){
      if(beam_noise == -1){
        //Calculating beam noise
        for(int j=0; j< fields[f].numVisibilitiesPerFreq[i]; j++){
            sum_inverse_weight += 1/fields[f].visibilities[i].weight[j];
        }
      }
      for(int j=0; j< fields[f].numVisibilitiesPerFreq[i]; j++){
          sum_weights += fields[f].visibilities[i].weight[j];
      }
  		fields[f].visibilities[i].numVisibilities = fields[f].numVisibilitiesPerFreq[i];
  		long UVpow2 = NearestPowerOf2(fields[f].visibilities[i].numVisibilities);
      fields[f].visibilities[i].threadsPerBlockUV = variables.blockSizeV;
  		fields[f].visibilities[i].numBlocksUV = UVpow2/fields[f].visibilities[i].threadsPerBlockUV;
    }
  }

  if(beam_noise == -1){
      beam_noise = sqrt(sum_inverse_weight)/total_visibilities;
      printf("Noise: %e\n", beam_noise);
  }

	if(num_gpus == 1){
    hipSetDevice(selected);
    for(int f=0; f<nfields; f++){
  		for(int i=0; i<data.total_frequencies; i++){
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].u, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].v, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
        gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vm, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vr, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  		}
    }
	}else{
    for(int f=0; f<nfields; f++){
  		for(int i=0; i<data.total_frequencies; i++){
  			hipSetDevice((i%num_gpus) + firstgpu);
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].u, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].v, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
        gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vm, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vr, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  		}
    }
	}


	if(num_gpus == 1){
    hipSetDevice(selected);
    for(int f=0; f<nfields; f++){
      gpuErrchk(hipMalloc((void**)&fields[f].atten_image, sizeof(hipfftComplex)*M*N));
      gpuErrchk(hipMemset(fields[f].atten_image, 0, sizeof(hipfftComplex)*M*N));
  		for(int i=0; i < data.total_frequencies; i++){

  			gpuErrchk(hipMalloc(&fields[f].device_vars[i].chi2, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMemset(fields[f].device_vars[i].chi2, 0, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));

  			gpuErrchk(hipMalloc((void**)&fields[f].device_vars[i].dchi2, sizeof(float3)*M*N));
  			gpuErrchk(hipMemset(fields[f].device_vars[i].dchi2, 0, sizeof(float3)*M*N));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].u, fields[f].visibilities[i].u, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].v, fields[f].visibilities[i].v, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].weight, fields[f].visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemset(fields[f].device_visibilities[i].Vr, 0, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
        gpuErrchk(hipMemset(fields[f].device_visibilities[i].Vm, 0, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));

  		}
    }
	}else{
    for(int f=0; f<nfields; f++){
      hipSetDevice(firstgpu);
      gpuErrchk(hipMalloc((void**)&fields[f].atten_image, sizeof(hipfftComplex)*M*N));
      gpuErrchk(hipMemset(fields[f].atten_image, 0, sizeof(hipfftComplex)*M*N));
  		for(int i=0; i < data.total_frequencies; i++){
  			hipSetDevice((i%num_gpus) + firstgpu);
        gpuErrchk(hipMalloc((void**)&fields[f].device_vars[i].device_S, sizeof(float)*M*N));
        gpuErrchk(hipMemset(fields[f].device_vars[i].device_S, 0, sizeof(float)*M*N));

  			gpuErrchk(hipMalloc(&fields[f].device_vars[i].chi2, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
  			gpuErrchk(hipMemset(fields[f].device_vars[i].chi2, 0, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));

  			gpuErrchk(hipMalloc((void**)&fields[f].device_vars[i].dchi2, sizeof(float3)*M*N));
  			gpuErrchk(hipMemset(fields[f].device_vars[i].dchi2, 0, sizeof(float3)*M*N));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].u, fields[f].visibilities[i].u, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].v, fields[f].visibilities[i].v, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].weight, fields[f].visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  			gpuErrchk(hipMemset(fields[f].device_visibilities[i].Vr, 0, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
        gpuErrchk(hipMemset(fields[f].device_visibilities[i].Vm, 0, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  		}
    }
	}

  //Declaring block size and number of blocks for Image
  dim3 threads(variables.blockSizeX, variables.blockSizeY);
	dim3 blocks(M/threads.x, N/threads.y);
	threadsPerBlockNN = threads;
	numBlocksNN = blocks;

	difmap_noise = beam_noise / (PI * beam_bmaj * beam_bmin / (4 * log(2) ));
  if(lambda == 0.0){
    MINPIX = 0.0;
  }else{
    if(reg_term == 0 && minpix == 0.0){
      printf("Cannot use entropy with a minimum value of pixel 0\n");
      goToError();
    }else{
      MINPIX = minpix;
    }
  }

	float deltax = RPDEG*DELTAX; //radians
	float deltay = RPDEG*DELTAY; //radians
	deltau = 1.0 / (M * deltax);
	deltav = 1.0 / (N * deltay);



	float3 *host_3I = (float3*)malloc(M*N*sizeof(float3));
  /////////////////////////////////////////////////////CALCULATE DIRECTION COSINES/////////////////////////////////////////////////
  double raimage = ra * RPDEG_D;
  double decimage = dec * RPDEG_D;
  if(verbose_flag){
    printf("FITS: Ra: %lf, dec: %lf\n", raimage, decimage);
  }
  for(int f=0; f<nfields; f++){
  	double lobs, mobs;

  	direccos(fields[f].obsra, fields[f].obsdec, raimage, decimage, &lobs,  &mobs);

  	fields[f].global_xobs = (crpix1 - 1.0) - lobs/deltax;
  	fields[f].global_yobs = (crpix2 - 1.0) - mobs/deltay;
    if(verbose_flag){
  	   printf("Field %d - Ra: %f, dec: %f , x0: %f, y0: %f\n", f, fields[f].obsra, fields[f].obsdec, fields[f].global_xobs, fields[f].global_yobs);
    }

    if(fields[f].global_xobs < 0 || fields[f].global_xobs > M || fields[f].global_xobs < 0 || fields[f].global_yobs > N) {
      printf("Pointing center (%f,%f) is outside the range of the image\n", fields[f].global_xobs, fields[f].global_xobs);
      goToError();
    }
  }
	////////////////////////////////////////////////////////MAKE STARTING IMAGE////////////////////////////////////////////////////////
	float *input_T = (float*)malloc(M*N*sizeof(float));
  int anynull;
  long fpixel = 1;
  float null = 0.;
  long elementsImage = M*N;

  if(strcmp(Tinput, "NULL")!=0){
    fitsfile *Tfile;
    int statusT = 0;
    fits_open_file(&Tfile, Tinput, 0, &statusT);
    fits_read_img(Tfile, TFLOAT, fpixel, elementsImage, &null, input_T, &anynull, &statusT);
  }

  int statustau = 0;
  float peak;
  float *input_tau= (float*)malloc(M*N*sizeof(float));
  fits_read_img(mod_in, TFLOAT, fpixel, elementsImage, &null, input_tau, &anynull, &statustau);
  peak = *std::max_element(input_tau,input_tau+(M*N));
  //fits_report_error(stderr, statustau); /* print error message */
  //printf("status: %d\n", statustau);
  int x = M-1;
  int y = N-1;
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
      if(strcmp(Tinput, "NULL")==0){
        host_3I[N*i+j].x = minpix_T;
      }else{
        host_3I[N*i+j].x = input_T[N*y+x];
      }
			//host_3I[N*i+j].x = peak/(1E26 * 2.0 * CBOLTZMANN * nu_0 * nu_0 / LIGHTSPEED * LIGHTSPEED); // T
			if(2.0*(input_tau[N*y+x]/peak) > minpix_tau){
			     host_3I[N*i+j].y = 2.0*(input_tau[N*y+x]/peak);  // tau
      }else{
        host_3I[N*i+j].y = minpix_tau;
      }
      host_3I[N*i+j].z = minpix_beta; // beta
      x--;
		}
    x=M-1;
    y--;
	}
  free(input_tau);
  free(input_T);
	////////////////////////////////////////////////CUDA MEMORY ALLOCATION FOR DEVICE///////////////////////////////////////////////////

	if(num_gpus == 1){
    hipSetDevice(selected);
    gpuErrchk(hipMalloc((void**)&device_Inu, sizeof(hipfftComplex)*M*N));
		gpuErrchk(hipMalloc((void**)&device_V, sizeof(hipfftComplex)*M*N));
	}else{
    for(int f = 0; f<nfields; f++){
  		for (int i = 0;  i < data.total_frequencies; i++) {
  			hipSetDevice((i%num_gpus) + firstgpu);
        gpuErrchk(hipMalloc((void**)&fields[f].device_vars[i].device_Inu, sizeof(hipfftComplex)*M*N));
  			gpuErrchk(hipMalloc((void**)&fields[f].device_vars[i].device_V, sizeof(hipfftComplex)*M*N));
  		}
    }
	}

  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
	   hipSetDevice(firstgpu);
  }
	gpuErrchk(hipMalloc((void**)&device_3I, sizeof(float3)*M*N));
  gpuErrchk(hipMemset(device_3I, 0, sizeof(float3)*M*N));

  gpuErrchk(hipMemcpy2D(device_3I, sizeof(float3), host_3I, sizeof(float3), sizeof(float3), M*N, hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**)&device_noise_image, sizeof(hipfftComplex)*M*N));
  gpuErrchk(hipMemset(device_noise_image, 0, sizeof(hipfftComplex)*M*N));

  gpuErrchk(hipMalloc((void**)&device_weight_image, sizeof(hipfftComplex)*M*N));
  gpuErrchk(hipMemset(device_weight_image, 0, sizeof(hipfftComplex)*M*N));

	gpuErrchk(hipMalloc((void**)&device_dphi, sizeof(float3)*M*N));
  gpuErrchk(hipMemset(device_dphi, 0, sizeof(float3)*M*N));

	gpuErrchk(hipMalloc((void**)&device_dchi2_total, sizeof(float3)*M*N));
  gpuErrchk(hipMemset(device_dchi2_total, 0, sizeof(float3)*M*N));


	gpuErrchk(hipMalloc((void**)&device_dS, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_dS, 0, sizeof(float)*M*N));

	gpuErrchk(hipMalloc((void**)&device_S, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_S, 0, sizeof(float)*M*N));



	if(num_gpus == 1){
    hipSetDevice(selected);
    gpuErrchk(hipMemset(device_Inu, 0, sizeof(hipfftComplex)*M*N));
		gpuErrchk(hipMemset(device_V, 0, sizeof(hipfftComplex)*M*N));
	}else{
    for(int f = 0; f < nfields; f++){
  		for (int i = 0;  i < data.total_frequencies; i++) {
  			hipSetDevice((i%num_gpus) + firstgpu);
  			gpuErrchk(hipMemset(fields[f].device_vars[i].device_V, 0, sizeof(hipfftComplex)*M*N));
  			gpuErrchk(hipMemset(fields[f].device_vars[i].device_Inu, 0, sizeof(hipfftComplex)*M*N));

  		}
    }
	}




	if(num_gpus == 1){
    hipSetDevice(selected);
		if ((hipfftPlan2d(&plan1GPU, N, M, HIPFFT_C2C))!= HIPFFT_SUCCESS) {
			printf("cufft plan error\n");
			return -1;
		}
	}else{
    for(int f = 0; f < nfields; f++){
  		for (int i = 0;  i < data.total_frequencies; i++) {
  			hipSetDevice((i%num_gpus) + firstgpu);
  			if ((hipfftPlan2d(&fields[f].device_vars[i].plan, N, M, HIPFFT_C2C))!= HIPFFT_SUCCESS) {
  				printf("cufft plan error\n");
  				return -1;
  			}
  		}
    }
	}

  //Time is taken from first kernel
  t = clock();
  start = omp_get_wtime();
	if(num_gpus == 1){
    hipSetDevice(selected);
    for(int f=0; f < nfields; f++){
  		for(int i=0; i<data.total_frequencies; i++){
  			hermitianSymmetry<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].u, fields[f].device_visibilities[i].v, fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].freq, fields[f].numVisibilitiesPerFreq[i]);
  			gpuErrchk(hipDeviceSynchronize());
  		}
    }
	}else{
    for(int f = 0; f < nfields; f++){
  		#pragma omp parallel for schedule(static,1)
      for (int i = 0; i < data.total_frequencies; i++)
  		{
  			unsigned int j = omp_get_thread_num();
  			//unsigned int num_cpu_threads = omp_get_num_threads();
  			// set and check the CUDA device for this CPU thread
  			int gpu_id = -1;
  			hipSetDevice((i%num_gpus) + firstgpu);   // "% num_gpus" allows more CPU threads than GPU devices
  			hipGetDevice(&gpu_id);
  			hermitianSymmetry<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].u, fields[f].device_visibilities[i].v, fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].freq, fields[f].numVisibilitiesPerFreq[i]);
  			gpuErrchk(hipDeviceSynchronize());
  		}

  	}
  }

	if(num_gpus == 1){
    hipSetDevice(selected);
    for(int f=0; f<nfields; f++){
  		for(int i=0; i<data.total_frequencies; i++){
        if(fields[f].numVisibilitiesPerFreq[i] > 0){
          total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(fields[f].atten_image, beam_fwhm, beam_freq, beam_cutoff, fields[f].visibilities[i].freq, fields[f].global_xobs, fields[f].global_yobs, DELTAX, DELTAY, N);
    			gpuErrchk(hipDeviceSynchronize());
        }
  		}
    }
	}else{
    for(int f=0; f<nfields; f++){
      #pragma omp parallel for schedule(static,1)
      for (int i = 0; i < data.total_frequencies; i++)
  		{
        unsigned int j = omp_get_thread_num();
  			//unsigned int num_cpu_threads = omp_get_num_threads();
  			// set and check the CUDA device for this CPU thread
  			int gpu_id = -1;
  			hipSetDevice((i%num_gpus) + firstgpu);   // "% num_gpus" allows more CPU threads than GPU devices
  			hipGetDevice(&gpu_id);
        if(fields[f].numVisibilitiesPerFreq[i] > 0){
    			#pragma omp critical
    			{
    				total_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(fields[f].atten_image, beam_fwhm, beam_freq, beam_cutoff, fields[f].visibilities[i].freq, fields[f].global_xobs, fields[f].global_yobs, DELTAX, DELTAY, N);
    				gpuErrchk(hipDeviceSynchronize());
    			}
        }
  		}
  	}
  }

  for(int f=0; f<nfields; f++){
    if(num_gpus == 1){
      hipSetDevice(selected);
    	mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(fields[f].atten_image, fields[f].valid_frequencies, N);
    	gpuErrchk(hipDeviceSynchronize());
  	}else{
      hipSetDevice(firstgpu);
    	mean_attenuation<<<numBlocksNN, threadsPerBlockNN>>>(fields[f].atten_image, fields[f].valid_frequencies, N);
    	gpuErrchk(hipDeviceSynchronize());
  	}
    toFitsFloat(fields[f].atten_image, f, M, N, 0);
  }

  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
	   hipSetDevice(firstgpu);
  }

  for(int f=0; f<nfields; f++){
    weight_image<<<numBlocksNN, threadsPerBlockNN>>>(device_weight_image, fields[f].atten_image, difmap_noise, N);
    gpuErrchk(hipDeviceSynchronize());
  }
  noise_image<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_weight_image, difmap_noise, N);
  gpuErrchk(hipDeviceSynchronize());
  toFitsFloat(device_noise_image, 0, M, N, 1);


	hipfftComplex *host_noise_image = (hipfftComplex*)malloc(M*N*sizeof(hipfftComplex));
	gpuErrchk(hipMemcpy2D(host_noise_image, sizeof(hipfftComplex), device_noise_image, sizeof(hipfftComplex), sizeof(hipfftComplex), M*N, hipMemcpyDeviceToHost));
	for(int i=0; i<M; i++){
		for(int j=0; j<N; j++){
			if(host_noise_image[N*i+j].x < noise_min){
				noise_min = host_noise_image[N*i+j].x;
			}
		}
	}

	fg_scale = noise_min;
	noise_cut = noise_cut * noise_min;
  if(verbose_flag){
	   printf("fg_scale = %e\n", fg_scale);
     printf("difmap_noise = %e\n", difmap_noise);
  }
	free(host_noise_image);
  hipFree(device_weight_image);
  for(int f=0; f<nfields; f++){
    hipFree(fields[f].atten_image);
  }



	//////////////////////////////////////////////////////Fletcher-Reeves Polak-Ribiere Minimization////////////////////////////////////////////////////////////////
	printf("\n\nStarting Fletcher Reeves Polak Ribiere method (Conj. Grad.)\n\n");
	float fret = 0.0;
	frprmn(device_3I	, ftol, &fret, chiCuadrado, dchiCuadrado);
  chiCuadrado(device_3I);
  t = clock() - t;
  end = omp_get_wtime();
  printf("Minimization ended successfully\n\n");
  printf("Iterations: %d\n", iter);
  printf("chi2: %f\n", final_chi2);
  printf("0.5*chi2: %f\n", 0.5*final_chi2);
  printf("Total visibilities: %d\n", total_visibilities);
  printf("Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
  printf("Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
  printf("S: %f\n", final_H);
  if(reg_term != 1){
    printf("Normalized S: %f\n", final_H/(M*N));
  }else{
    printf("Normalized S: %f\n", final_H/(M*M*N*N));
  }
  printf("lambda*S: %f\n\n", lambda*final_H);
	double time_taken = ((double)t)/CLOCKS_PER_SEC;
  double wall_time = end-start;
  printf("Total CPU time: %lf\n", time_taken);
  printf("Wall time: %lf\n\n\n", wall_time);

  if(strcmp(variables.ofile,"NULL") != 0){
    FILE *outfile = fopen(variables.ofile, "w");
    if (outfile == NULL)
    {
        printf("Error opening output file!\n");
        goToError();
    }

    fprintf(outfile, "Iterations: %d\n", iter);
    fprintf(outfile, "chi2: %f\n", final_chi2);
    fprintf(outfile, "0.5*chi2: %f\n", 0.5*final_chi2);
    fprintf(outfile, "Total visibilities: %d\n", total_visibilities);
    fprintf(outfile, "Reduced-chi2 (Num visibilities): %f\n", (0.5*final_chi2)/total_visibilities);
    fprintf(outfile, "Reduced-chi2 (Weights sum): %f\n", (0.5*final_chi2)/sum_weights);
    fprintf(outfile, "S: %f\n", final_H);
    if(reg_term != 1){
      fprintf(outfile, "Normalized S: %f\n", final_H/(M*N));
    }else{
      fprintf(outfile, "Normalized S: %f\n", final_H/(M*M*N*N));
    }
    fprintf(outfile, "lambda*S: %f\n", lambda*final_H);
    fprintf(outfile, "Wall time: %lf", wall_time);
    fclose(outfile);
  }
	//Pass residuals to host
	printf("Saving final image to disk\n");
	float3toImage(device_3I, freqavg, iter, M, N, 0);
	//Saving residuals to disk
  residualsToHost(fields, data);
  printf("Saving residuals to MS...\n");
	writeMS(msinput,msoutput,fields);
	printf("Residuals saved.\n");

	//Free device and host memory
	printf("Free device and host memory\n");
	hipfftDestroy(plan1GPU);
  for(int f=0; f<nfields; f++){
  	for(int i=0; i<data.total_frequencies; i++){
      if(num_gpus > 1){
  		    hipSetDevice((i%num_gpus) + firstgpu);
      }
  		hipFree(fields[f].device_visibilities[i].u);
  		hipFree(fields[f].device_visibilities[i].v);
  		hipFree(fields[f].device_visibilities[i].weight);

  		hipFree(fields[f].device_visibilities[i].Vr);
  		hipFree(fields[f].device_visibilities[i].Vo);

  		hipfftDestroy(fields[f].device_vars[i].plan);
  	}
  }

  for(int f=0; f<nfields; f++){
  	for(int i=0; i<data.total_frequencies; i++){
      if(fields[f].numVisibilitiesPerFreq[i] != 0){
    		free(fields[f].visibilities[i].u);
    		free(fields[f].visibilities[i].v);
    		free(fields[f].visibilities[i].weight);
    		free(fields[f].visibilities[i].Vo);
        free(fields[f].visibilities[i].Vm);
      }
  	}
  }

	hipFree(device_3I);
	if(num_gpus == 1){
		hipFree(device_V);
		hipFree(device_Inu);
	}else{
    for(int f=0; f<nfields;f++){
  		for(int i=0; i<data.total_frequencies; i++){
  			hipSetDevice((i%num_gpus) + firstgpu);
  			hipFree(fields[f].device_vars[i].device_V);
  			hipFree(fields[f].device_vars[i].device_Inu);
        hipFree(fields[f].device_vars[i].device_S);
  		}
    }
	}
  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
    hipSetDevice(firstgpu);
  }

	hipFree(device_noise_image);

	hipFree(device_dphi);
	hipFree(device_dchi2_total);
	hipFree(device_dS);

	hipFree(device_chi2);
	hipFree(device_S);

  //Disabling UVA
  if(num_gpus > 1){
    for(int i=firstgpu+1; i<num_gpus+firstgpu; i++){
          hipSetDevice(firstgpu);
          hipDeviceDisablePeerAccess(i);
          hipSetDevice(i);
          hipDeviceDisablePeerAccess(firstgpu);
    }

    for(int i=0; i<num_gpus; i++ ){
          hipSetDevice((i%num_gpus) + firstgpu);
          hipDeviceReset();
    }
  }
	free(host_3I);
	free(msinput);
	free(msoutput);
	free(modinput);

  fits_close_file(mod_in, &status_mod_in);
  if (status_mod_in) {
    fits_report_error(stderr, status_mod_in);
    goToError();
  }

	return 0;
}
