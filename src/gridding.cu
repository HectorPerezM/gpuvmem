#include "hip/hip_runtime.h"
#include "gridding.cuh"

extern double deltau, deltav;
extern float robust_param;
extern long M, N;
extern int num_gpus;

Gridding::Gridding()
{
        this->threads = 1;
};

void Gridding::applyCriteria(Visibilities *v)
{
        for(int f=0; f< v->getData()->nfields; f++) {
                for(int i=0; i < v->getData()->total_frequencies; i++) {
                    for(int s=0; i < v->getData()->nstokes; s++) {
                        v->getFields()[f].gridded_visibilities[i][s].uvw = (double3 *) malloc(M * N * sizeof(double3));
                        v->getFields()[f].gridded_visibilities[i][s].weight = (float *) malloc(M * N * sizeof(float));
                        v->getFields()[f].gridded_visibilities[i][s].Vo = (hipfftComplex *) malloc(
                                M * N * sizeof(hipfftComplex));

                        memset(v->getFields()[f].gridded_visibilities[i][s].uvw, 0, M * N * sizeof(double3));
                        memset(v->getFields()[f].gridded_visibilities[i][s].weight, 0, M * N * sizeof(float));
                        memset(v->getFields()[f].gridded_visibilities[i][s].S, 0, M * N * sizeof(int));
                        memset(v->getFields()[f].gridded_visibilities[i][s].Vo, 0, M * N * sizeof(hipfftComplex));
                    }
                }
        }
        omp_set_num_threads(threads);
        do_gridding(v->getFields(), v->getData(), deltau, deltav, M, N, robust_param);
        omp_set_num_threads(num_gpus);
};

Gridding::Gridding(int threads)
{
        if(threads != 1 && threads >= 1)
                this->threads = threads;
        else if(threads != 1)
                printf("Number of threads set to 1\n");
};

void Gridding::configure(void *params)
{
        int *threads = (int*) params;
        printf("Number of threads = %d\n", *threads);
        if(*threads != 1 && *threads >= 1)
                this->threads = *threads;
        else if(*threads != 1)
                printf("Number of threads set to 1\n");
};

namespace {
Filter* CreateGridding()
{
        return new Gridding;
}
const int GriddingId = 0;
const bool RegisteredGridding = Singleton<FilterFactory>::Instance().RegisterFilter(GriddingId, CreateGridding);
};
