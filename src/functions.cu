#include "hip/hip_runtime.h"
#include "functions.cuh"


extern long M, N;
extern int numVisibilities, iterations, iterthreadsVectorNN, blocksVectorNN, nopositivity, crpix1, crpix2, \
status_mod_in, verbose_flag, xcorr_flag, nsamples, nstokes, num_gpus, selected, iter;

extern hipfftHandle plan1GPU;
extern hipfftComplex *device_I, *device_V, *device_noise_image, *device_fg_image, *device_image;

extern float *device_dphi, *device_chi2, *device_H, *device_dchi2_total, *device_dH;
extern float difmap_noise, fg_scale, global_xobs, global_yobs, DELTAX, DELTAY, deltau, deltav, noise_cut, MINPIX, \
minpix_factor, lambda, ftol, random_probability, final_chi2, final_H;

extern dim3 threadsPerBlockNN, numBlocksNN;

extern float beam_noise, beam_bmaj, beam_bmin, b_noise_aux;
extern double ra, dec, obsra, obsdec;

extern freqData data;
extern VPF *device_vars;
extern Vis *visibilities, *device_visibilities;


extern char* mempath, *out_image;

extern fitsfile *mod_in;

__host__ void goToError()
{
  for(int i=1; i<num_gpus; i++){
        hipSetDevice(0);
        hipDeviceDisablePeerAccess(i);
        hipSetDevice(i%num_gpus);
        hipDeviceDisablePeerAccess(0);
  }

  for(int i=0; i<num_gpus; i++ ){
        hipSetDevice(i%num_gpus);
        hipDeviceReset();
  }

  printf("An error has ocurred, exiting\n");
  exit(0);

}

__host__ freqData getFreqs(char * file)
{
   freqData freqsAndVisibilities;
   string dir = file;
   string query;
   casa::Vector<double> pointing;
   casa::Table main_tab(dir);
   casa::Table field_tab(main_tab.keywordSet().asTable("FIELD"));
   casa::Table spectral_window_tab(main_tab.keywordSet().asTable("SPECTRAL_WINDOW"));
   casa::Table polarization_tab(main_tab.keywordSet().asTable("POLARIZATION"));
   int fields = field_tab.nrow();
   //For now only 1 FIELD.
   casa::ROTableRow field_row(field_tab, casa::stringToVector("REFERENCE_DIR,NAME"));
   const casa::TableRecord &values = field_row.get(0);
   pointing = values.asArrayDouble("REFERENCE_DIR");
   obsra = pointing[0];
   obsdec = pointing[1];

   nsamples = main_tab.nrow();
   if (nsamples == 0) {
      printf("ERROR : nsamples is zero... exiting....\n");
      exit(-1);
   }

  casa::ROArrayColumn<casa::Double> chan_freq_col(spectral_window_tab,"CHAN_FREQ");  //NUMBER OF SPW
  freqsAndVisibilities.n_internal_frequencies = spectral_window_tab.nrow();

  freqsAndVisibilities.channels = (int*)malloc(freqsAndVisibilities.n_internal_frequencies*sizeof(int));
  casa::ROScalarColumn<casa::Int> n_chan_freq(spectral_window_tab,"NUM_CHAN");
  for(int i = 0; i < freqsAndVisibilities.n_internal_frequencies; i++){
    freqsAndVisibilities.channels[i] = n_chan_freq(i);
  }

  int total_frequencies = 0;
  for(int i=0; i <freqsAndVisibilities.n_internal_frequencies; i++){
    for(int j=0; j < freqsAndVisibilities.channels[i]; j++){
      total_frequencies++;
    }
  }

  freqsAndVisibilities.total_frequencies = total_frequencies;
  freqsAndVisibilities.numVisibilitiesPerFreq = (long*)malloc(freqsAndVisibilities.total_frequencies*sizeof(long));
  for(int i=0;i<freqsAndVisibilities.total_frequencies;i++){
    freqsAndVisibilities.numVisibilitiesPerFreq[i] = 0;
  }

  casa::ROScalarColumn<casa::Int> n_corr(polarization_tab,"NUM_CORR");
  nstokes=n_corr(0);

  casa::ROTableRow row(main_tab, casa::stringToVector("FLAG,FLAG_ROW,FIELD_ID,UVW,WEIGHT,SIGMA,ANTENNA1,ANTENNA2,TIME,EXPOSURE,DATA,DATA_DESC_ID"));
  casa::Vector<casa::Bool> auxbool;
  bool flag;
  int spw;
  int counter = 0;
  for(int i=0; i < freqsAndVisibilities.n_internal_frequencies; i++){
    for(int j=0; j < freqsAndVisibilities.channels[i]; j++){
      for (int k=0; k < nsamples; k++){
        const casa::TableRecord &values = row.get(k);
        flag = values.asBool("FLAG_ROW");
        spw = values.asInt("DATA_DESC_ID");
        casa::Array<casa::Bool> flagCol = values.asArrayBool("FLAG");
        if(spw == i && flag == false){
          for (int sto=0; sto<nstokes; sto++){
            auxbool = flagCol[j][sto];
            if(auxbool[0] == false){
              freqsAndVisibilities.numVisibilitiesPerFreq[counter]++;
            }
          }
        }else continue;
      }
      counter++;
    }
  }

  return freqsAndVisibilities;
}

__host__ long NearestPowerOf2(long n)
{
  if (!n) return n;  //(0 == 2^0)

  int x = 1;
  while(x < n)
  {
      x <<= 1;
  }
  return x;
}


bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}


__host__ void readInputDat(char *file)
{
  FILE *fp;
  char item[50];
  float status;
  if((fp = fopen(file, "r")) == NULL){
    printf("ERROR. The input file wasn't provided by the user.\n");
    goToError();
  }else{
    while(true){
      int ret = fscanf(fp, "%s %e", item, &status);

      if(ret==EOF){
        break;
      }else{
        if (strcmp(item,"lambda_entropy")==0) {
          if(lambda == -1){
            lambda = status;
          }
        }else if (strcmp(item,"noise_cut")==0){
          noise_cut = status;
        }else if(strcmp(item,"minpix_factor")==0){
          minpix_factor = status;
        } else if(strcmp(item,"ftol")==0){
          ftol = status;
        } else if(strcmp(item,"random_probability")==0){
          random_probability = status;
        }else{
          break;
        }
      }
    }
  }
}
__host__ void readMS(char *file, char *file2, Vis *visibilities) {
  ///////////////////////////////////////////////////FITS READING///////////////////////////////////////////////////////////
  status_mod_in = 0;
  int status_noise = 0;
  fits_open_file(&mod_in, file2, 0, &status_mod_in);
  if (status_mod_in) {
    fits_report_error(stderr, status_mod_in); /* print error message */
    goToError();
  }


  fits_read_key(mod_in, TFLOAT, "CDELT1", &DELTAX, NULL, &status_mod_in);
  fits_read_key(mod_in, TFLOAT, "CDELT2", &DELTAY, NULL, &status_mod_in);
  fits_read_key(mod_in, TDOUBLE, "CRVAL1", &ra, NULL, &status_mod_in);
  fits_read_key(mod_in, TDOUBLE, "CRVAL2", &dec, NULL, &status_mod_in);
  fits_read_key(mod_in, TINT, "CRPIX1", &crpix1, NULL, &status_mod_in);
  fits_read_key(mod_in, TINT, "CRPIX2", &crpix2, NULL, &status_mod_in);
  fits_read_key(mod_in, TLONG, "NAXIS1", &M, NULL, &status_mod_in);
  fits_read_key(mod_in, TLONG, "NAXIS2", &N, NULL, &status_mod_in);
  fits_read_key(mod_in, TFLOAT, "BMAJ", &beam_bmaj, NULL, &status_mod_in);
  fits_read_key(mod_in, TFLOAT, "BMIN", &beam_bmin, NULL, &status_mod_in);
  fits_read_key(mod_in, TFLOAT, "NOISE", &beam_noise, NULL, &status_noise);
  if (status_mod_in) {
    fits_report_error(stderr, status_mod_in); /* print error message */
    goToError();
  }

  if(status_noise){
    beam_noise = b_noise_aux;
  }

  beam_bmaj = beam_bmaj/-DELTAX;
  beam_bmin = beam_bmin/-DELTAX;

  if(verbose_flag){
    printf("FITS Files READ\n");
  }

  ///////////////////////////////////////////////////MS SQLITE READING/////////////////////////////////////////////////////////
  char *error = 0;
  int g = 0, h = 0;
  string query;
  string dir = file;
  casa::Table main_tab(dir);
  casa::Table field_tab(main_tab.keywordSet().asTable("FIELD"));
  casa::Table spectral_window_tab(main_tab.keywordSet().asTable("SPECTRAL_WINDOW"));
  casa::Table polarization_tab(main_tab.keywordSet().asTable("POLARIZATION"));

  casa::ROArrayColumn<casa::Int> correlation_col(polarization_tab,"CORR_TYPE");
  casa::Vector<int> polarizations;
  polarizations=correlation_col(0);

  casa::ROArrayColumn<casa::Double> chan_freq_col(spectral_window_tab,"CHAN_FREQ");

  casa::ROTableRow row(main_tab, casa::stringToVector("FLAG,FLAG_ROW,FIELD_ID,UVW,WEIGHT,SIGMA,ANTENNA1,ANTENNA2,TIME,EXPOSURE,DATA,DATA_DESC_ID"));
  casa::Vector<casa::Bool> auxbool;
  casa::Vector<float> v;
  casa::Vector<float> weights;
  casa::Vector<double> uvw;
  bool flag;
  int spw;

  if(random_probability != 0.0){
    float u;
    SelectStream(0);
    PutSeed(1);
    for(int i=0; i < data.n_internal_frequencies; i++){
      for(int j=0; j < data.channels[i]; j++){
        for (int k=0; k < nsamples; k++){
          const casa::TableRecord &values = row.get(k);
          uvw = values.asArrayDouble("UVW");
          flag = values.asBool("FLAG_ROW");
          spw = values.asInt("DATA_DESC_ID");
          casa::Array<casa::Complex> dataCol = values.asArrayComplex ("DATA");
          casa::Array<casa::Bool> flagCol = values.asArrayBool("FLAG");
          weights=values.asArrayFloat ("WEIGHT");
          if(spw == i && flag == false){
            for (int sto=0; sto<nstokes; sto++){
              auxbool = flagCol[j][sto];
              if(auxbool[0] == false){
                u = Random();
                if(u<1-random_probability){
                  visibilities[g].stokes[h] = polarizations[sto];
                  visibilities[g].u[h] = uvw[0];
                  visibilities[g].v[h] = uvw[1];
                  v = casa::real(dataCol[j][sto]);
                  visibilities[g].Vo[h].x = v[0];
                  v = casa::imag(dataCol[j][sto]);
                  visibilities[g].Vo[h].y = v[0];
                  visibilities[g].weight[h] = weights[sto];
                  h++;
                }
              }
            }
          }else continue;
        }
        data.numVisibilitiesPerFreq[g] = (h+1);
        realloc(visibilities[g].stokes, (h+1)*sizeof(int));
        realloc(visibilities[g].u, (h+1)*sizeof(float));
        realloc(visibilities[g].v, (h+1)*sizeof(float));
        realloc(visibilities[g].Vo, (h+1)*sizeof(hipfftComplex));
        realloc(visibilities[g].weight, (h+1)*sizeof(float));
        h=0;
        g++;
      }
    }
  }else{
    for(int i=0; i < data.n_internal_frequencies; i++){
      for(int j=0; j < data.channels[i]; j++){
        for (int k=0; k < nsamples; k++){
          const casa::TableRecord &values = row.get(k);
          uvw = values.asArrayDouble("UVW");
          flag = values.asBool("FLAG_ROW");
          spw = values.asInt("DATA_DESC_ID");
          casa::Array<casa::Complex> dataCol = values.asArrayComplex("DATA");
          casa::Array<casa::Bool> flagCol = values.asArrayBool("FLAG");
          weights=values.asArrayFloat("WEIGHT");
          if(spw == i && flag == false){
            for (int sto=0; sto<nstokes; sto++) {
              auxbool = flagCol[j][sto];
              if(auxbool[0] == false){
                visibilities[g].stokes[h] = polarizations[sto];
                visibilities[g].u[h] = uvw[0];
                visibilities[g].v[h] = uvw[1];
                v = casa::real(dataCol[j][sto]);
                visibilities[g].Vo[h].x = v[0];
                v = casa::imag(dataCol[j][sto]);
                visibilities[g].Vo[h].y = v[0];
                visibilities[g].weight[h] = weights[sto];
                h++;
              }
            }
          }else continue;
        }
        h=0;
        g++;
      }
    }
  }


  h = 0;
  for(int i = 0; i < data.n_internal_frequencies; i++){
    casa::Vector<double> chan_freq_vector;
    chan_freq_vector=chan_freq_col(i);
    for(int j = 0; j < data.channels[i]; j++){
      visibilities[h].freq = chan_freq_vector[j];
      h++;
    }
  }


}

__host__ void MScopy(char const *in_dir, char const *in_dir_dest) {
  string dir_origin=in_dir;
  string dir_dest=in_dir_dest;

  casa::Table tab_src(dir_origin);
  tab_src.deepCopy(dir_dest,casa::Table::New);
  if (verbose_flag) {
      printf("Copied\n");
  }


}



__host__ void residualsToHost(Vis *device_visibilities, Vis *visibilities, freqData data){
  printf("Saving residuals to host memory\n");
  if(num_gpus == 1){
    for(int i=0; i<data.total_frequencies; i++){
      gpuErrchk(hipMemcpy(visibilities[i].Vr, device_visibilities[i].Vr, sizeof(hipfftComplex)*data.numVisibilitiesPerFreq[i], hipMemcpyDeviceToHost));
    }
  }else{
    for(int i=0; i<data.total_frequencies; i++){
      hipSetDevice(i%num_gpus);
      gpuErrchk(hipMemcpy(visibilities[i].Vr, device_visibilities[i].Vr, sizeof(hipfftComplex)*data.numVisibilitiesPerFreq[i], hipMemcpyDeviceToHost));
    }
  }

  for(int i=0; i<data.total_frequencies; i++){
    for(int j=0; j<data.numVisibilitiesPerFreq[i];j++){
      if(visibilities[i].u[j]<0){
        visibilities[i].Vr[j].y *= -1;
      }
    }
  }

}

__host__ void writeMS(char *infile, char *outfile, Vis *visibilities) {
  MScopy(infile, outfile);
  char* out_col = "DATA";
  string dir=outfile;
  string query;
  casa::Table main_tab(dir,casa::Table::Update);
  string column_name=out_col;

  if (main_tab.tableDesc().isColumn(column_name))
  {
    printf("Column %s already exists... skipping creation...\n", out_col);
  }else{
    printf("Adding %s to the main table...\n", out_col);
    main_tab.addColumn(casa::ArrayColumnDesc <casa::Complex>(column_name,"created by gpuvmem"));
    main_tab.flush();
  }

  if (column_name!="DATA")
  {
     query="UPDATE "+dir+" set "+column_name+"=DATA";
     printf("Duplicating DATA column into %s\n", out_col);
     casa::tableCommand(query);
  }

  casa::TableRow row(main_tab, casa::stringToVector(column_name+",FLAG,FLAG_ROW,DATA_DESC_ID"));
  casa::Complex comp;
  casa::Vector<casa::Bool> auxbool;
  bool flag;
  int spw, h = 0, g = 0;
  hipfftComplex before;
  for(int i=0; i < data.n_internal_frequencies; i++){
    for(int j=0; j < data.channels[i]; j++){
      for (int k=0; k < nsamples; k++){
        const casa::TableRecord &values = row.get(k);
        flag = values.asBool("FLAG_ROW");
        spw = values.asInt("DATA_DESC_ID");
        casa::Array<casa::Bool> flagCol = values.asArrayBool("FLAG");
        casa::Array<casa::Complex> dataCol = values.asArrayComplex(column_name);
        if(spw == i && flag == false){
          for (int sto=0; sto<nstokes; sto++){
            auxbool = flagCol[j][sto];
            if(auxbool[0] == false){
              comp.real() = -visibilities[g].Vr[h].x;
              comp.imag() = -visibilities[g].Vr[h].y;
              printf("Saving Re:%f, Im:%f, spw: %d, sample: %d in dataCol[%d][%d]\n", -visibilities[g].Vr[h].x, -visibilities[g].Vr[h].y, i, k, j, sto);
              if(j>=1){
                printf("The value stored in dataCol[%d][%d] in sample: %d , spw: %d, is: %f, %f\n", j-1, sto, k, i, casa::real(dataCol[j-1][sto]), casa::imag(dataCol[j-1][sto]));
              }
              dataCol[j][sto] = comp;
              h++;
            }
          }
          row.put(k);
        }else continue;
      }
      h=0;
      g++;
    }
  }
  main_tab.flush();

}

__host__ void print_help() {
	printf("Example: ./bin/gpuvmem options [ arguments ...]\n");
	printf("    -h  --help       Shows this\n");
  printf(	"   -X  --blockSizeX       Block X Size for Image (Needs to be pow of 2)\n");
  printf(	"   -Y  --blockSizeY       Block Y Size for Image (Needs to be pow of 2)\n");
  printf(	"   -V  --blockSizeV       Block Size for Visibilities (Needs to be pow of 2)\n");
  printf(	"   -i  --input       The name of the input file of visibilities(MS)\n");
  printf(	"   -o  --output       The name of the output file of residual visibilities(MS)\n");
  printf(	"   -O  --output-image       The name of the output image FITS file\n");
  printf("    -I  --inputdat       The name of the input file of parameters\n");
  printf("    -m  --modin       mod_in_0 FITS file\n");
  printf("    -n  --noise       Noise Parameter (Optional)\n");
  printf("    -l  --lambda      Lambda Regulatization Parameter (Optional)\n");
  printf("    -p  --path       MEM folder path to save FITS images. With last / included. (Example ./../mem/)\n");
  printf("    -M  --multigpu       Number of GPUs to use multiGPU image synthesis (Default OFF => 0)\n");
  printf("    -s  --select       If multigpu option is OFF, then select the GPU ID of the GPU you will work on. (Default = 0)\n");
  printf("    -t  --iterations       Number of iterations for optimization (Default = 50)\n");
  printf("        --xcorr       Run gpuvmem with cross-correlation\n");
  printf("        --nopositivity       Run gpuvmem using chi2 with no posititivy restriction\n");
  printf("        --verbose       Shows information through all the execution\n");
}

__host__ char *strip(const char *string, const char *chars)
{
  char * newstr = (char*)malloc(strlen(string) + 1);
  int counter = 0;

  for ( ; *string; string++) {
    if (!strchr(chars, *string)) {
      newstr[ counter ] = *string;
      ++ counter;
    }
  }

  newstr[counter] = 0;
  return newstr;
}

__host__ Vars getOptions(int argc, char **argv) {
	Vars variables;
  variables.multigpu = 0;
  variables.select = 0;
  variables.blockSizeX = -1;
  variables.blockSizeY = -1;
  variables.blockSizeV = -1;
  variables.it_max = 500;
  variables.noise = -1;
  variables.lambda = -1;


	long next_op;
	const char* const short_op = "hi:o:O:I:m:n:l:M:s:p:X:Y:V:t:";

	const struct option long_op[] = { //Flag for help
                                    {"help", 0, NULL, 'h' },
                                    /* These options set a flag. */
                                    {"verbose", 0, &verbose_flag, 1},
                                    {"xcorr", 0, &xcorr_flag, 1},
                                    {"nopositivity", 0, &nopositivity, 1},
                                    /* These options don’t set a flag. */
                                    {"input", 1, NULL, 'i' }, {"output", 1, NULL, 'o'}, {"output-image", 1, NULL, 'O'},
                                    {"inputdat", 1, NULL, 'I'}, {"modin", 1, NULL, 'm' }, {"noise", 0, NULL, 'n' },
                                    {"lambda", 0, NULL, 'l' }, {"multigpu", 0, NULL, 'M'}, {"select", 1, NULL, 's'},
                                    {"path", 1, NULL, 'p'}, {"blockSizeX", 1, NULL, 'X'}, {"blockSizeY", 1, NULL, 'Y'},
                                    {"blockSizeV", 1, NULL, 'V'}, {"iterations", 0, NULL, 't'}, { NULL, 0, NULL, 0 }};

	if (argc == 1) {
		printf(
				"ERROR. THE PROGRAM HAS BEEN EXECUTED WITHOUT THE NEEDED PARAMETERS OR OPTIONS\n");
		print_help();
		exit(EXIT_SUCCESS);
	}
  int option_index = 0;
	while (1) {
		next_op = getopt_long(argc, argv, short_op, long_op, &option_index);
		if (next_op == -1) {
			break;
		}

		switch (next_op) {
    case 0:
      /* If this option set a flag, do nothing else now. */
      if (long_op[option_index].flag != 0)
        break;
        printf ("option %s", long_op[option_index].name);
      if (optarg)
        printf (" with arg %s", optarg);
        printf ("\n");
        break;
		case 'h':
			print_help();
			exit(EXIT_SUCCESS);
		case 'i':
      variables.input = (char*) malloc((strlen(optarg)+1)*sizeof(char));
			strcpy(variables.input, optarg);
			break;
    case 'o':
      variables.output = (char*) malloc((strlen(optarg)+1)*sizeof(char));
  		strcpy(variables.output, optarg);
  		break;
    case 'O':
      variables.output_image = (char*) malloc((strlen(optarg)+1)*sizeof(char));
    	strcpy(variables.output_image, optarg);
    	break;
    case 'I':
      variables.inputdat = (char*) malloc((strlen(optarg)+1)*sizeof(char));
      strcpy(variables.inputdat, optarg);
      break;
    case 'm':
      variables.modin = (char*) malloc((strlen(optarg)+1)*sizeof(char));
    	strcpy(variables.modin, optarg);
    	break;
    case 'n':
      variables.noise = atof(optarg);
      break;
    case 'l':
      variables.lambda = atof(optarg);
      break;
    case 'p':
      variables.path = (char*) malloc((strlen(optarg)+1)*sizeof(char));
      strcpy(variables.path, optarg);
      break;
    case 'M':
      variables.multigpu = atoi(optarg);
      break;
    case 's':
      variables.select = atoi(optarg);
      break;
    case 'X':
      variables.blockSizeX = atoi(optarg);
      break;
    case 'Y':
      variables.blockSizeY = atoi(optarg);
      break;
    case 'V':
      variables.blockSizeV = atoi(optarg);
      break;
    case 't':
      variables.it_max = atoi(optarg);
      break;
		case '?':
			print_help();
			exit(EXIT_FAILURE);
		case -1:
			break;
		default:
      print_help();
			exit(EXIT_FAILURE);
		}
	}

  if(variables.blockSizeX == -1 && variables.blockSizeY == -1 && variables.blockSizeV == -1 ||
     strip(variables.input, " ") == "" && strip(variables.output, " ") == "" && strip(variables.output_image, " ") == "" && strip(variables.inputdat, " ") == "" ||
     strip(variables.modin, " ") == "" && strip(variables.path, " ") == "") {
        print_help();
        exit(EXIT_FAILURE);
  }

  if(!isPow2(variables.blockSizeX) && !isPow2(variables.blockSizeY) && !isPow2(variables.blockSizeV)){
    print_help();
    exit(EXIT_FAILURE);
  }

  if(variables.multigpu != 0 && variables.select != 0){
    print_help();
    exit(EXIT_FAILURE);
  }
	return variables;
}

__host__ void toFitsFloat(hipfftComplex *I, int iteration, long M, long N, int option)
{
	fitsfile *fpointer;
	int status = 0;
	long fpixel = 1;
	long elements = M*N;
	char name[60]="";
	long naxes[2]={M,N};
	long naxis = 2;
  char *unit = "JY/PIXEL";
  switch(option){
    case 0:
      sprintf(name, "!%s", out_image);
      break;
    case 1:
      sprintf(name, "!%sMEM_%d.fits", mempath, iteration);
      break;
    case 2:
      sprintf(name, "!%sMEM_V_%d.fits", mempath, iteration);
      break;
    case 3:
      sprintf(name, "!%sMEM_VB_%d.fits", mempath, iteration);
      break;
    case 4:
      sprintf(name, "!%satten_%d.fits", mempath, iteration);
      break;
    case 5:
      sprintf(name, "!%stotal_atten_0.fits", mempath, iteration);
      break;
    case 6:
      sprintf(name, "!%snoise_0.fits", mempath, iteration);
      break;
    case -1:
      break;
    default:
      printf("Invalid case to FITS\n");
      goToError();
  }


	fits_create_file(&fpointer, name, &status);
  if (status) {
    fits_report_error(stderr, status); /* print error message */
    goToError();
  }
  fits_copy_header(mod_in, fpointer, &status);
  if (status) {
    fits_report_error(stderr, status); /* print error message */
    goToError();
  }
  if(option==0){
    fits_update_key(fpointer, TSTRING, "BUNIT", unit, "Unit of measurement", &status);
  }
  hipfftComplex *host_IFITS;
  host_IFITS = (hipfftComplex*)malloc(M*N*sizeof(hipfftComplex));
  gpuErrchk(hipMemcpy2D(host_IFITS, sizeof(hipfftComplex), I, sizeof(hipfftComplex), sizeof(hipfftComplex), M*N, hipMemcpyDeviceToHost));

	float* image2D;
	image2D = (float*) malloc(M*N*sizeof(float));

  int x = M-1;
  int y = N-1;
  for(int i=0; i < M; i++){
		for(int j=0; j < N; j++){
      if(option == 0){
			  image2D[N*y+x] = host_IFITS[N*i+j].x * fg_scale;
      }else if (option == 2 || option == 3){
        image2D[N*y+x] = sqrt(host_IFITS[N*i+j].x * host_IFITS[N*i+j].x + host_IFITS[N*i+j].y * host_IFITS[N*i+j].y);
        //image2D[N*x+y] = host_IFITS[N*i+j].y;
      }else if(option == 4 || option == 5 || option == 6){
        image2D[N*i+j] = host_IFITS[N*i+j].x;
      }else{
        image2D[N*y+x] = host_IFITS[N*i+j].x;
      }
      x--;
		}
    x=M-1;
    y--;
	}

	fits_write_img(fpointer, TFLOAT, fpixel, elements, image2D, &status);
  if (status) {
    fits_report_error(stderr, status); /* print error message */
    goToError();
  }
	fits_close_file(fpointer, &status);
  if (status) {
    fits_report_error(stderr, status); /* print error message */
    goToError();
  }

  free(host_IFITS);
	free(image2D);
}


template <bool nIsPow2>
__global__ void deviceReduceKernel(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int blockSize = blockDim.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    float mySum = 0.f;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}





__host__ float deviceReduce(float *in, long N) {
  float *device_out;
  gpuErrchk(hipMalloc(&device_out, sizeof(float)*1024));
  gpuErrchk(hipMemset(device_out, 0, sizeof(float)*1024));

  int threads = 512;
  int blocks = min((int(NearestPowerOf2(N)) + threads - 1) / threads, 1024);
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

  bool isPower2 = isPow2(N);
  if(isPower2){
    deviceReduceKernel<true><<<blocks, threads, smemSize>>>(in, device_out, N);
    gpuErrchk(hipDeviceSynchronize());
  }else{
    deviceReduceKernel<false><<<blocks, threads, smemSize>>>(in, device_out, N);
    gpuErrchk(hipDeviceSynchronize());
  }

  float *h_odata = (float *) malloc(blocks*sizeof(float));
  float sum = 0.0;

  gpuErrchk(hipMemcpy(h_odata, device_out, blocks * sizeof(float),hipMemcpyDeviceToHost));
  for (int i=0; i<blocks; i++)
  {
    sum += h_odata[i];
  }
  hipFree(device_out);
  free(h_odata);
	return sum;
}

__global__ void hermitianSymmetry(float *Ux, float *Vx, hipfftComplex *Vo, float freq, int numVisibilities)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < numVisibilities){
      if(Ux[i] < 0.0){
        Ux[i] *= -1.0;
        Vx[i] *= -1.0;
        Vo[i].y *= -1.0;
      }
      Ux[i] = (Ux[i] * freq) / LIGHTSPEED;
      Vx[i] = (Vx[i] * freq) / LIGHTSPEED;
  }
}

__global__ void attenuation(hipfftComplex *attenMatrix, float frec, long N, float xobs, float yobs, float DELTAX, float DELTAY)
{

		int j = threadIdx.x + blockDim.x * blockIdx.x;
		int i = threadIdx.y + blockDim.y * blockIdx.y;

    int x0 = xobs;
    int y0 = yobs;
    float x = (j - x0) * DELTAX * RPDEG;
    float y = (i - y0) * DELTAY * RPDEG;

    float arc = sqrtf(x*x+y*y);
    float c = 4.0*logf(2.0);
    //printf("frec:%f\n", frec);
    float a = (FWHM*BEAM_FREQ/(frec*1e-9));
    float r = arc/a;
    float atten = expf(-c*r*r);
    attenMatrix[N*i+j].x = atten;
    attenMatrix[N*i+j].y = 0;
}



__global__ void total_attenuation(hipfftComplex *total_atten, hipfftComplex *attenperFreq, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;

  total_atten[N*i+j].x += attenperFreq[N*i+j].x;
  total_atten[N*i+j].y = 0;
}

__global__ void mean_attenuation(hipfftComplex *total_atten, int channels, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;

  total_atten[N*i+j].x /= channels;
  total_atten[N*i+j].y = 0;
}

__global__ void noise_image(hipfftComplex *total_atten, hipfftComplex *noise_image, float difmap_noise, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;

  float weight = 0.0;
  float noiseval = 0.0;
  float atten = total_atten[N*i+j].x;
  weight = (atten / difmap_noise) * (atten / difmap_noise);
  noiseval = sqrtf(1.0/weight);
  noise_image[N*i+j].x = noiseval;
  noise_image[N*i+j].y = 0;
}

__global__ void apply_beam(hipfftComplex *image, hipfftComplex *fg_image, long N, float xobs, float yobs, float fg_scale, float frec, float DELTAX, float DELTAY)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;


    float dx = DELTAX * 60.0;
    float dy = DELTAY * 60.0;
    float x = (j - xobs) * dx;
    float y = (i - yobs) * dy;
    float arc = RPARCM*sqrtf(x*x+y*y);
    float c = 4.0*logf(2.0);
    float a = (FWHM*BEAM_FREQ/(frec*1e-9));
    float r = arc/a;
    float atten = expf(-c*r*r);

    image[N*i+j].x = fg_image[N*i+j].x * fg_scale * atten;
    image[N*i+j].y = 0.f;
}

/*--------------------------------------------------------------------
 * Phase rotate the visibility data in "image" to refer phase to point
 * (x,y) instead of (0,0).
 * Multiply pixel V(i,j) by exp(-2 pi i (x/ni + y/nj))
 *--------------------------------------------------------------------*/
__global__ void phase_rotate(hipfftComplex *data, long M, long N, float xphs, float yphs)
{

		int j = threadIdx.x + blockDim.x * blockIdx.x;
		int i = threadIdx.y + blockDim.y * blockIdx.y;

    float u,v;
    float du = xphs/M;
    float dv = yphs/N;

    if(j < M/2){
      u = du * j;
    }else{
      u = du * (j-M);
    }

    if(i < N/2){
      v = dv * i;
    }else{
      v = dv * (i-N);
    }

    float phase = -2.0*(u+v);
    float c, s;
    #if (__CUDA_ARCH__ >= 300 )
      sincospif(phase, &s, &c);
    #else
      c = cospif(phase);
      s = sinpif(phase);
    #endif
    float  re = data[N*i+j].x;
    float im = data[N*i+j].y;
    data[N*i+j].x = re * c - im * s;
    data[N*i+j].y = re * s + im * c;
}


/*
 * Interpolate in the visibility array to find the visibility at (u,v);
 */
__global__ void vis_mod(hipfftComplex *Vm, hipfftComplex *Vo, hipfftComplex *V, float *Ux, float *Vx, float deltau, float deltav, long numVisibilities, long N)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  long i1, i2, j1, j2;
  float du, dv;
  float v11, v12, v21, v22;
  float Zreal;
  float Zimag;
  if (i < numVisibilities){
    float u = Ux[i]/deltau;
    float v = Vx[i]/deltav;

    if (fabsf(u) > (N/2)+0.5 || fabsf(v) > (N/2)+0.5) {
      printf("Error in residual: u,v = %f,%f\n", u, v);
      asm("trap;");
    }

    if(u < 0.0){
      u = N + u;
    }

    if(v < 0.0){
      v = N + v;
    }

    i1 = u;
    i2 = (i1+1)%N;
    du = u - i1;
    j1 = v;
    j2 = (j1+1)%N;
    dv = v - j1;

    if (i1 < 0 || i1 > N || j1 < 0 || j2 > N) {
      printf("Error in residual: u,v = %f,%f, %ld,%ld, %ld,%ld\n", u, v, i1, i2, j1, j2);
      asm("trap;");
    }

    /*if(i1 == 511 || i2 == 0){
      printf("Positions (%ld,%ld); (%ld, %ld); (%ld, %ld); (%ld, %ld)\n", i1, j1, i1, j2, i2, j1, i2, j2);
    }*/
    /* Bilinear interpolation: real part */
    v11 = V[N*j1 + i1].x; /* [i1, j1] */
    v12 = V[N*j2 + i1].x; /* [i1, j2] */
    v21 = V[N*j1 + i2].x; /* [i2, j1] */
    v22 = V[N*j2 + i2].x; /* [i2, j2] */
    Zreal = (1-du)*(1-dv)*v11 + (1-du)*dv*v12 + du*(1-dv)*v21 + du*dv*v22;
    /* Bilinear interpolation: imaginary part */
    v11 = V[N*j1 + i1].y; /* [i1, j1] */
    v12 = V[N*j2 + i1].y; /* [i1, j2] */
    v21 = V[N*j1 + i2].y; /* [i2, j1] */
    v22 = V[N*j2 + i2].y; /* [i2, j2] */
    Zimag = (1-du)*(1-dv)*v11 + (1-du)*dv*v12 + du*(1-dv)*v21 + du*dv*v22;

    Vm[i].x = Zreal;
    Vm[i].y = Zimag;

  }

}

__global__ void alphaVectors(float *alpha_num, float *alpha_den, float *w, hipfftComplex *Vm, hipfftComplex *Vo, long numVisibilities){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < numVisibilities){
    alpha_num[i] = w[i] * ((Vm[i].x * Vo[i].x) + (Vm[i].y * Vo[i].y));
    alpha_den[i] = w[i] * ((Vm[i].x * Vm[i].x) + (Vm[i].y * Vm[i].y));
  }

}

__global__ void residual(hipfftComplex *Vr, hipfftComplex *Vm, hipfftComplex *Vo, long numVisibilities){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < numVisibilities){
    Vr[i].x = Vm[i].x - Vo[i].x;
    Vr[i].y = Vm[i].y - Vo[i].y;
  }
}


__global__ void residual_XCORR(hipfftComplex *Vr, hipfftComplex *Vm, hipfftComplex *Vo, float alpha, long numVisibilities){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < numVisibilities){
    Vr[i].x = (alpha * Vm[i].x) - Vo[i].x;
    Vr[i].y = (alpha * Vm[i].y) - Vo[i].y;
  }
}

__global__ void clipWNoise(hipfftComplex *fg_image, hipfftComplex *noise, hipfftComplex *I, long N, float noise_cut, float MINPIX)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;


  if(noise[N*i+j].x > noise_cut){
    I[N*i+j].x = MINPIX;
  }

  fg_image[N*i+j].x = I[N*i+j].x;
  //printf("%f\n", fg_image[N*i+j].x);
  fg_image[N*i+j].y = 0;
}


__global__ void getGandDGG(float *gg, float *dgg, float *xi, float *g, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  gg[N*i+j] = g[N*i+j] * g[N*i+j];
  dgg[N*i+j] = (xi[N*i+j] + g[N*i+j]) * xi[N*i+j];
}

__global__ void newP(hipfftComplex *p, float *xi, float xmin, float MINPIX, long N)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  xi[N*i+j] *= xmin;
  if(p[N*i+j].x + xi[N*i+j] > MINPIX){
    p[N*i+j].x += xi[N*i+j];
  }else{
    p[N*i+j].x = MINPIX;
    xi[N*i+j] = 0.0;
  }
  p[N*i+j].y = 0.0;
}

__global__ void newPNoPositivity(hipfftComplex *p, float *xi, float xmin, long N)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  xi[N*i+j] *= xmin;
  p[N*i+j].x += xi[N*i+j];
  p[N*i+j].y = 0.0;
}

__global__ void evaluateXt(hipfftComplex *xt, hipfftComplex *pcom, float *xicom, float x, float MINPIX, long N)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  if(pcom[N*i+j].x + x * xicom[N*i+j] > MINPIX){
    xt[N*i+j].x = pcom[N*i+j].x + x * xicom[N*i+j];
  }else{
      xt[N*i+j].x = MINPIX;
  }
  xt[N*i+j].y = 0.0;
}

__global__ void evaluateXtNoPositivity(hipfftComplex *xt, hipfftComplex *pcom, float *xicom, float x, long N)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  xt[N*i+j].x = pcom[N*i+j].x + x * xicom[N*i+j];
  xt[N*i+j].y = 0.0;
}


__global__ void chi2Vector(float *chi2, hipfftComplex *Vr, float *w, long numVisibilities){
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < numVisibilities){
		chi2[i] =  w[i] * ((Vr[i].x * Vr[i].x) + (Vr[i].y * Vr[i].y));
	}

}

__global__ void HVector(float *H, hipfftComplex *noise, hipfftComplex *I, long N, float noise_cut, float MINPIX)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  float entropy = 0.0;
  if(noise[N*i+j].x <= noise_cut){
    entropy = I[N*i+j].x * logf(I[N*i+j].x / MINPIX);
  }

  H[N*i+j] = entropy;
}

__global__ void QVector(float *H, hipfftComplex *noise, hipfftComplex *I, long N, float noise_cut, float MINPIX)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  float entropy = 0.0;
  if(noise[N*i+j].x <= noise_cut){
    if((i>0 && i<N) && (j>0 && j<N)){
      entropy = (I[N*i+j].x - I[N*i+(j-1)].x) * (I[N*i+j].x - I[N*i+(j-1)].x) + (I[N*i+j].x - I[N*i+(j+1)].x) * (I[N*i+j].x - I[N*i+(j+1)].x) + (I[N*i+j].x - I[N*(i-1)+j].x) * (I[N*i+j].x - I[N*(i-1)+j].x) + (I[N*i+j].x - I[N*(i+1)+j].x) * (I[N*i+j].x - I[N*(i+1)+j].x);
      entropy /= 2;
    }else{
      entropy = I[N*i+j].x;
    }
  }

  H[N*i+j] = entropy;
}

__global__ void TVVector(float *TV, hipfftComplex *noise, hipfftComplex *I, long N, float noise_cut, float MINPIX)
{
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  float tv = 0.0;
  if(noise[N*i+j].x <= noise_cut){
    if(i!= N-1 || j!=N-1){
      float dx = I[N*i+(j+1)].x - I[N*i+j].x;
      float dy = I[N*(i+1)+j].x - I[N*i+j].x;
      tv = sqrtf((dx * dx) + (dy * dy));
    }else{
      tv = 0;
    }
  }

  TV[N*i+j] = tv;
}
__global__ void searchDirection(float *g, float *xi, float *h, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  g[N*i+j] = -xi[N*i+j];
  xi[N*i+j] = h[N*i+j] = g[N*i+j];
}

__global__ void newXi(float *g, float *xi, float *h, float gam, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  g[N*i+j] = -xi[N*i+j];
  xi[N*i+j] = h[N*i+j] = g[N*i+j] + gam * h[N*i+j];
}

__global__ void restartDPhi(float *dphi, float *dChi2, float *dH, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  int i = threadIdx.y + blockDim.y * blockIdx.y;

  dphi[N*i+j] = 0.0;
  dChi2[N*i+j] = 0.0;
  dH[N*i+j] = 0.0;

}

__global__ void DH(float *dH, hipfftComplex *I, hipfftComplex *noise, float noise_cut, float lambda, float MINPIX, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  if(noise[N*i+j].x <= noise_cut){
    dH[N*i+j] = lambda * (logf(I[N*i+j].x / MINPIX) + 1.0);
  }
}

__global__ void DQ(float *dQ, hipfftComplex *I, hipfftComplex *noise, float noise_cut, float lambda, float MINPIX, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  if(noise[N*i+j].x <= noise_cut){
    if((i>0 && i<N) && (j>0 && j<N)){
    //dQ[N*i+j] = lambda * (logf(I[N*i+j].x / MINPIX) + 1.0);
    dQ[N*i+j] = (I[N*i+j].x - I[N*i+(j-1)].x) + (I[N*i+j].x - I[N*i+(j+1)].x) + (I[N*i+j].x - I[N*(i-1)+j].x)  + (I[N*i+j].x - I[N*(i+1)+j].x);
  }else{
    dQ[N*i+j] = I[N*i+j].x;
    }
    dQ[N*i+j] *= lambda;
  }
}

__global__ void DTV(float *dTV, hipfftComplex *I, hipfftComplex *noise, float noise_cut, float lambda, float MINPIX, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  float dtv = 0.0;
  float num = 0.0;
  float den = 0.0;
  if(noise[N*i+j].x <= noise_cut){
    if(i!= N-1 || j!=N-1){
      float a = I[N*i+(j+1)].x;
      float b = I[N*(i+1)+j].x;
      float y = I[N*i+j].x;
      float num = -a-b+(2*y);
      float den = (a*a) - 2*y*(a+b) + (b*b) + 2*(y*y);
      if(den <= 0){
        dtv = MINPIX;
      }else{
        dtv = num/sqrtf(den);
      }
    }else{
      dtv = MINPIX;
    }
    dTV[N*i+j] = lambda * dtv;
  }
}

__global__ void DChi2(hipfftComplex *noise, hipfftComplex *atten, float *dChi2, hipfftComplex *Vr, float *U, float *V, float *w, long N, long numVisibilities, float fg_scale, float noise_cut, float xobs, float yobs, float DELTAX, float DELTAY)
{

	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  int x0 = xobs;
  int y0 = yobs;
  float x = (j - x0) * DELTAX * RPDEG;
  float y = (i - y0) * DELTAY * RPDEG;

	float Ukv;
	float Vkv;

	float cosk;
	float sink;

  float dchi2 = 0.0;
  if(noise[N*i+j].x <= noise_cut){
  	for(int v=0; v<numVisibilities; v++){
      Ukv = x * U[v];
  		Vkv = y * V[v];
      #if (__CUDA_ARCH__ >= 300 )
        sincospif(2.0*(Ukv+Vkv), &sink, &cosk);
      #else
        cosk = cospif(2.0*(Ukv+Vkv));
        sink = sinpif(2.0*(Ukv+Vkv));
      #endif
      dchi2 += w[v]*((Vr[v].x * cosk) - (Vr[v].y * sink));
  	}

  dchi2 *= fg_scale * atten[N*i+j].x;
  dChi2[N*i+j] = dchi2;
  }
}


__global__ void DChi2_XCORR(hipfftComplex *noise, hipfftComplex *atten, float *dChi2, hipfftComplex *Vr, float *U, float *V, float *w, long N, long numVisibilities, float fg_scale, float noise_cut, float xobs, float yobs, float alpha, float DELTAX, float DELTAY)
{

	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  int x0 = xobs;
  int y0 = yobs;
  float x = (j - x0) * DELTAX * RPDEG;
  float y = (i - y0) * DELTAY * RPDEG;

	float Ukv;
	float Vkv;

	float cosk;
	float sink;

  float dchi2 = 0.0;
  if(noise[N*i+j].x <= noise_cut){
  	for(int v=0; v<numVisibilities; v++){
      Ukv = x * U[v];
  		Vkv = y * V[v];
      #if (__CUDA_ARCH__ >= 300 )
        sincospif(2.0*(Ukv+Vkv), &sink, &cosk);
      #else
        cosk = cospif(2.0*(Ukv+Vkv));
        sink = sinpif(2.0*(Ukv+Vkv));
      #endif
      dchi2 += w[v]*((Vr[v].x * cosk) - (Vr[v].y * sink));
  	}

  dchi2 *= alpha * fg_scale * atten[N*i+j].x;
  dChi2[N*i+j] = dchi2;
  }
}

__global__ void DChi2_total(float *dchi2_total, float *dchi2, long N)
{

	int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  dchi2_total[N*i+j] += dchi2[N*i+j];
}

__global__ void DPhi(float *dphi, float *dchi2, float *dH, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  dphi[N*i+j] = dchi2[N*i+j] + dH[N*i+j];
}

__global__ void substraction(float *x, hipfftComplex *xc, float *gc, float lambda, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  x[N*i+j] = xc[N*i+j].x - lambda*gc[N*i+j];
}

__global__ void projection(float *px, float *x, float MINPIX, long N){

  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;


  if(INFINITY < x[N*i+j]){
    px[N*i+j] = INFINITY;
  }else{
    px[N*i+j] = x[N*i+j];
  }

  if(MINPIX > px[N*i+j]){
    px[N*i+j] = MINPIX;
  }else{
    px[N*i+j] = px[N*i+j];
  }
}

__global__ void normVectorCalculation(float *normVector, float *gc, long N){
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  normVector[N*i+j] = gc[N*i+j] * gc[N*i+j];
}

__global__ void copyImage(hipfftComplex *p, float *device_xt, long N)
{
  int j = threadIdx.x + blockDim.x * blockIdx.x;
	int i = threadIdx.y + blockDim.y * blockIdx.y;

  p[N*i+j].x = device_xt[N*i+j];
}


__host__ float chiCuadrado(hipfftComplex *I)
{
  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
    hipSetDevice(0);
  }

  float resultPhi = 0.0;
  float resultchi2  = 0.0;
  float resultH  = 0.0;


  clipWNoise<<<numBlocksNN, threadsPerBlockNN>>>(device_fg_image, device_noise_image, I, N, noise_cut, MINPIX);
  gpuErrchk(hipDeviceSynchronize());


  if(iter>0 && MINPIX!=0.0){
    HVector<<<numBlocksNN, threadsPerBlockNN>>>(device_H, device_noise_image, device_fg_image, N, noise_cut, MINPIX);
    gpuErrchk(hipDeviceSynchronize());
  }

  if(num_gpus == 1){
    hipSetDevice(selected);
    for(int i=0; i<data.total_frequencies;i++){

    	apply_beam<<<numBlocksNN, threadsPerBlockNN>>>(device_image, device_fg_image, N, global_xobs, global_yobs, fg_scale, visibilities[i].freq, DELTAX, DELTAY);
    	gpuErrchk(hipDeviceSynchronize());

    	//FFT 2D
    	if ((hipfftExecC2C(plan1GPU, (hipfftComplex*)device_image, (hipfftComplex*)device_V, HIPFFT_FORWARD)) != HIPFFT_SUCCESS) {
    		printf("CUFFT exec error\n");
    		goToError();
    	}
    	gpuErrchk(hipDeviceSynchronize());

      //PHASE_ROTATE VISIBILITIES
      phase_rotate<<<numBlocksNN, threadsPerBlockNN>>>(device_V, M, N, global_xobs, global_yobs);
    	gpuErrchk(hipDeviceSynchronize());

      //RESIDUAL CALCULATION
      vis_mod<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_visibilities[i].Vm, device_visibilities[i].Vo, device_V, device_visibilities[i].u, device_visibilities[i].v, deltau, deltav, data.numVisibilitiesPerFreq[i], N);
    	gpuErrchk(hipDeviceSynchronize());

      if(xcorr_flag==1 && iter>1){
        float alpha_num = 1.0;
        float alpha_den = 1.0;
        alphaVectors<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_vars[i].alpha_num, device_vars[i].alpha_den, device_visibilities[i].weight, device_visibilities[i].Vm, device_visibilities[i].Vo, data.numVisibilitiesPerFreq[i]);

        alpha_num = deviceReduce(device_vars[i].alpha_num, data.numVisibilitiesPerFreq[i]);

        alpha_den = deviceReduce(device_vars[i].alpha_den, data.numVisibilitiesPerFreq[i]);

        if(alpha_den == 0.0){
          alpha_num = 1.0;
          alpha_den = 1.0;
        }

        device_vars[i].alpha = alpha_num/alpha_den;

        residual_XCORR<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_visibilities[i].Vr, device_visibilities[i].Vm, device_visibilities[i].Vo, device_vars[i].alpha, data.numVisibilitiesPerFreq[i]);
      }else{
        residual<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_visibilities[i].Vr, device_visibilities[i].Vm, device_visibilities[i].Vo, data.numVisibilitiesPerFreq[i]);
      }

    	////chi 2 VECTOR
    	chi2Vector<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_vars[i].chi2, device_visibilities[i].Vr, device_visibilities[i].weight, data.numVisibilitiesPerFreq[i]);
    	gpuErrchk(hipDeviceSynchronize());

    	//REDUCTIONS
    	//chi2
    	resultchi2  += deviceReduce(device_vars[i].chi2, data.numVisibilitiesPerFreq[i]);
    }
  }else{
    #pragma omp parallel for schedule(static,1)
    for (int i = 0; i < data.total_frequencies; i++)
		{
      float result = 0.0;
      unsigned int j = omp_get_thread_num();
			//unsigned int num_cpu_threads = omp_get_num_threads();
			// set and check the CUDA device for this CPU thread
			int gpu_id = -1;
			hipSetDevice(i % num_gpus);   // "% num_gpus" allows more CPU threads than GPU devices
			hipGetDevice(&gpu_id);

    	apply_beam<<<numBlocksNN, threadsPerBlockNN>>>(device_vars[i].device_image, device_fg_image, N, global_xobs, global_yobs, fg_scale, visibilities[i].freq, DELTAX, DELTAY);
    	gpuErrchk(hipDeviceSynchronize());

    	//FFT 2D
    	if ((hipfftExecC2C(device_vars[i].plan, (hipfftComplex*)device_vars[i].device_image, (hipfftComplex*)device_vars[i].device_V, HIPFFT_FORWARD)) != HIPFFT_SUCCESS) {
    		printf("CUFFT exec error\n");
    		//return -1 ;
    		goToError();
    	}
    	gpuErrchk(hipDeviceSynchronize());

      //PHASE_ROTATE VISIBILITIES
      phase_rotate<<<numBlocksNN, threadsPerBlockNN>>>(device_vars[i].device_V, M, N, global_xobs, global_yobs);
    	gpuErrchk(hipDeviceSynchronize());

      //RESIDUAL CALCULATION
      vis_mod<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_visibilities[i].Vm, device_visibilities[i].Vo, device_vars[i].device_V, device_visibilities[i].u, device_visibilities[i].v, deltau, deltav, data.numVisibilitiesPerFreq[i], N);
    	gpuErrchk(hipDeviceSynchronize());


      if(xcorr_flag==1 && iter>1){
        float alpha_num = 1.0;
        float alpha_den = 1.0;
        alphaVectors<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_vars[i].alpha_num, device_vars[i].alpha_den, device_visibilities[i].weight, device_visibilities[i].Vm, device_visibilities[i].Vo, data.numVisibilitiesPerFreq[i]);

        alpha_num = deviceReduce(device_vars[i].alpha_num, data.numVisibilitiesPerFreq[i]);

        alpha_den = deviceReduce(device_vars[i].alpha_den, data.numVisibilitiesPerFreq[i]);

        if(alpha_den == 0.0){
          alpha_num = 1.0;
          alpha_den = 1.0;
        }

        device_vars[i].alpha = alpha_num/alpha_den;

        residual_XCORR<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_visibilities[i].Vr, device_visibilities[i].Vm, device_visibilities[i].Vo, device_vars[i].alpha, data.numVisibilitiesPerFreq[i]);
      }else{
        residual<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_visibilities[i].Vr, device_visibilities[i].Vm, device_visibilities[i].Vo, data.numVisibilitiesPerFreq[i]);
      }


    	////chi2 VECTOR
    	chi2Vector<<<visibilities[i].numBlocksUV, visibilities[i].threadsPerBlockUV>>>(device_vars[i].chi2, device_visibilities[i].Vr, device_visibilities[i].weight, data.numVisibilitiesPerFreq[i]);
    	gpuErrchk(hipDeviceSynchronize());


      result = deviceReduce(device_vars[i].chi2, data.numVisibilitiesPerFreq[i]);
    	//REDUCTIONS
    	//chi2
      #pragma omp critical
      {
        resultchi2  += result;
      }

    }
  }
  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
    hipSetDevice(0);
  }
  resultH  = deviceReduce(device_H, M*N);
  resultPhi = (0.5 * resultchi2) + (lambda * resultH);

  final_chi2 = resultchi2;
  final_H = resultH;
  /*printf("chi2 value = %.5f\n", resultchi2);
  printf("H value = %.5f\n", resultH);
  printf("(1/2) * chi2 value = %.5f\n", 0.5*resultchi2);
  printf("lambda * H value = %.5f\n", lambda*resultH);
  printf("Phi value = %.5f\n\n", resultPhi);*/

  return resultPhi;
}



__host__ void dchiCuadrado(hipfftComplex *I, float *dxi2)
{

  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
    hipSetDevice(0);
  }

  restartDPhi<<<numBlocksNN, threadsPerBlockNN>>>(device_dphi, device_dchi2_total, device_H, N);
  gpuErrchk(hipDeviceSynchronize());


  toFitsFloat(I, iter, M, N, 1);
  //toFitsFloat(device_V, iter, M, N, 2);

  if(iter>0 && MINPIX!=0.0){

    DH<<<numBlocksNN, threadsPerBlockNN>>>(device_dH, I, device_noise_image, noise_cut, lambda, MINPIX, N);
    gpuErrchk(hipDeviceSynchronize());

  }

  if(num_gpus == 1){
    hipSetDevice(selected);
    for(int i=0; i<data.total_frequencies;i++){
        if(xcorr_flag==1 && iter>1){
          DChi2_XCORR<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_vars[i].atten, device_vars[i].dchi2, device_visibilities[i].Vr, device_visibilities[i].u, device_visibilities[i].v, device_visibilities[i].weight, N, data.numVisibilitiesPerFreq[i], fg_scale, noise_cut, global_xobs, global_yobs, device_vars[i].alpha, DELTAX, DELTAY);
        	gpuErrchk(hipDeviceSynchronize());
        }else{
          DChi2<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_vars[i].atten, device_vars[i].dchi2, device_visibilities[i].Vr, device_visibilities[i].u, device_visibilities[i].v, device_visibilities[i].weight, N, data.numVisibilitiesPerFreq[i], fg_scale, noise_cut, global_xobs, global_yobs, DELTAX, DELTAY);
        	gpuErrchk(hipDeviceSynchronize());
        }


        DChi2_total<<<numBlocksNN, threadsPerBlockNN>>>(device_dchi2_total, device_vars[i].dchi2, N);
      	gpuErrchk(hipDeviceSynchronize());

    }
  }else{
    #pragma omp parallel for schedule(static,1)
    for (int i = 0; i < data.total_frequencies; i++)
    {
      unsigned int j = omp_get_thread_num();
      //unsigned int num_cpu_threads = omp_get_num_threads();
      // set and check the CUDA device for this CPU thread
      int gpu_id = -1;
      hipSetDevice(i % num_gpus);   // "% num_gpus" allows more CPU threads than GPU devices
      hipGetDevice(&gpu_id);
      if(xcorr_flag==1 && iter>1){
        DChi2_XCORR<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_vars[i].atten, device_vars[i].dchi2, device_visibilities[i].Vr, device_visibilities[i].u, device_visibilities[i].v, device_visibilities[i].weight, N, data.numVisibilitiesPerFreq[i], fg_scale, noise_cut, global_xobs, global_yobs, device_vars[i].alpha, DELTAX, DELTAY);
        gpuErrchk(hipDeviceSynchronize());
      }else{
        DChi2<<<numBlocksNN, threadsPerBlockNN>>>(device_noise_image, device_vars[i].atten, device_vars[i].dchi2, device_visibilities[i].Vr, device_visibilities[i].u, device_visibilities[i].v, device_visibilities[i].weight, N, data.numVisibilitiesPerFreq[i], fg_scale, noise_cut, global_xobs, global_yobs, DELTAX, DELTAY);
        gpuErrchk(hipDeviceSynchronize());
      }

      #pragma omp critical
      {
        DChi2_total<<<numBlocksNN, threadsPerBlockNN>>>(device_dchi2_total, device_vars[i].dchi2, N);
        gpuErrchk(hipDeviceSynchronize());
      }

    }
  }

  if(num_gpus == 1){
    hipSetDevice(selected);
  }else{
    hipSetDevice(0);
  }

  DPhi<<<numBlocksNN, threadsPerBlockNN>>>(device_dphi, device_dchi2_total, device_dH, N);
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy2D(dxi2, sizeof(float), device_dphi, sizeof(float), sizeof(float), M*N, hipMemcpyDeviceToDevice));

}
