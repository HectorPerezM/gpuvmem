#include "linmin.cuh"
#define TOL 1.0e-7

hipfftComplex *device_pcom;
float *device_xicom, (*nrfunc)(hipfftComplex*);
extern long M;
extern long N;
extern float MINPIX;
extern int positivity;

extern dim3 threadsPerBlockNN;
extern dim3 numBlocksNN;
extern int verbose_flag;


__host__ void linmin(hipfftComplex *p, float *xi, float *fret, float (*func)(hipfftComplex*))//p and xi are in GPU
{
  float xx, xmin, fx, fb, fa, bx ,ax;

  gpuErrchk(hipMalloc((void**)&device_pcom, sizeof(hipfftComplex)*M*N));
  gpuErrchk(hipMemset(device_pcom, 0, sizeof(hipfftComplex)*M*N));

  gpuErrchk((hipMalloc((void**)&device_xicom, sizeof(float)*M*N)));
  gpuErrchk(hipMemset(device_xicom, 0, sizeof(float)*M*N));
  nrfunc = func;
  //device_pcom = p;
  //device_xicom = xi;
  gpuErrchk(hipMemcpy2D(device_pcom, sizeof(hipfftComplex), p, sizeof(hipfftComplex), sizeof(hipfftComplex), M*N, hipMemcpyDeviceToDevice));
  gpuErrchk(hipMemcpy2D(device_xicom, sizeof(float), xi, sizeof(float), sizeof(float), M*N, hipMemcpyDeviceToDevice));

  ax = 0.0;
	xx = 1.0;

  mnbrak(&ax, &xx, &bx, &fa, &fx, &fb, f1dim);


  *fret = brent(ax, xx, bx, TOL, &xmin, f1dim);
  if(verbose_flag){
    printf("alpha = %f\n\n", xmin);
  }

  //GPU MUL AND ADD
  //xi     = xi*xmin;
  //p      = p + xi;
  if(positivity == 1){
    newP<<<numBlocksNN, threadsPerBlockNN>>>(p, xi, xmin, MINPIX, N);
    gpuErrchk(hipDeviceSynchronize());
  }else{
    newPNoPositivity<<<numBlocksNN, threadsPerBlockNN>>>(p, xi, xmin, N);
    gpuErrchk(hipDeviceSynchronize());
  }

  hipFree(device_xicom);
  hipFree(device_pcom);
}
#undef TOL
