#include "imageProcessor.cuh"

extern int image_count;
extern long N, M;

ImageProcessor::ImageProcessor()
{
}

void ImageProcessor::clip(float *I)
{
  if(image_count == 1)
  {
    linkClip(I);
  }

  if(image_count == 2)
  {
    linkClip(I);
  }
};
void ImageProcessor::calculateInu(hipfftComplex *image, float *I, float freq)
{
  if(image_count == 2)
  {
    linkCalculateInu2I(image, I, freq);
  }
};

void ImageProcessor::apply_beam(hipfftComplex *image, float xobs, float yobs, float freq)
{
  if(image_count == 1)
    linkApplyBeam1I(image, fg_image, xobs, yobs, freq);
  if(image_count == 2)
    linkApplyBeam2I(image, xobs, yobs, freq);
};

void ImageProcessor::chainRule(float *I, float freq)
{
  if(image_count == 2)
  {
    linkChain2I(chain, freq, I);
  }

};

void ImageProcessor::clip(hipfftComplex *I)
{
  if(image_count == 1)
  {

  }
};

void ImageProcessor::clipWNoise(float *I)
{
  if(image_count == 1)
    linkClipWNoise1I(fg_image, I);
  if(image_count == 2)
    linkClipWNoise2I(I);
};

void ImageProcessor::configure(int I)
{
  this->image_count = I;
  if(image_count == 1)
  {
    gpuErrchk(hipMalloc((void**)&fg_image, sizeof(hipfftComplex)*M*N));
    gpuErrchk(hipMemset(fg_image, 0, sizeof(hipfftComplex)*M*N));
  }

  if(image_count > 1)
  {
    gpuErrchk(hipMalloc((void**)&chain, sizeof(float)*M*N*image_count));
    gpuErrchk(hipMemset(chain, 0, sizeof(float)*M*N*image_count));
  }
};
