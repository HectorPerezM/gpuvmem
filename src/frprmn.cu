#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
  Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus,
  Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl

  This program includes Numerical Recipes (NR) based routines whose
  copyright is held by the NR authors. If NR routines are included,
  you are required to comply with the licensing set forth there.

	Part of the program also relies on an an ANSI C library for multi-stream
	random number generation from the related Prentice-Hall textbook
	Discrete-Event Simulation: A First Course by Steve Park and Larry Leemis,
  for more information please contact leemis@math.wm.edu

  Additionally, this program uses some NVIDIA routines whose copyright is held
  by NVIDIA end user license agreement (EULA).

  For the original parts of this code, the following license applies:

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program. If not, see <http://www.gnu.org/licenses/>.
* -------------------------------------------------------------------------
*/

#include "frprmn.cuh"

extern long M;
extern long N;
extern int iter;

extern dim3 threadsPerBlockNN;
extern dim3 numBlocksNN;

extern int threadsVectorNN;
extern int blocksVectorNN;

extern float MINPIX;
extern int verbose_flag;

#define EPS 1.0e-10
extern int it_maximum;

#define ARMIJOTOLERANCE 1.e-6
#define ALPHA 1.e-4


#define FREEALL hipFree(device_gg_vector);hipFree(device_dgg_vector);hipFree(xi);hipFree(device_h);hipFree(device_g);
#define FREEALL_LBFGS hipFree(d_y);hipFree(d_s);hipFree(xi);hipFree(xi_old);hipFree(p_old);hipFree(norm_vector);


__host__ void armijoTest(hipfftComplex *p, float (*func)(hipfftComplex*), void (*dfunc)(hipfftComplex*, float*))
{
  int i = 0;
  double start, end;
  int iarm;
  float normPGC, normPL, fc, ft, fgoal;
  float *device_normVector, *device_pgc, *device_x, *xi, *device_xt, *device_pl;

  gpuErrchk(hipMalloc((void**)&device_normVector, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_normVector, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_pgc, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_pgc, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_x, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_x, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&xi, sizeof(float)*M*N));
  gpuErrchk(hipMemset(xi, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_xt, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_xt, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_pl, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_pl, 0, sizeof(float)*M*N));


  fc = (*func)(p);
  printf("Function value = %f\n", fc);
  //exit(0);
  (*dfunc)(p,xi);

  substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_x, p, xi, 1.0, N);
  gpuErrchk(hipDeviceSynchronize());

  projection<<<numBlocksNN, threadsPerBlockNN>>>(device_xt, device_x, MINPIX, N);
  gpuErrchk(hipDeviceSynchronize());

  substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_pgc, p, device_xt, 1.0, N);
  gpuErrchk(hipDeviceSynchronize());

  normVectorCalculation<<<numBlocksNN, threadsPerBlockNN>>>(device_normVector, device_pgc, N);
  gpuErrchk(hipDeviceSynchronize());

  normPGC = deviceReduce<float>(device_normVector, M*N);
  i=1;
  while(normPGC > ARMIJOTOLERANCE && i <= it_maximum){
    start = omp_get_wtime();
    iter = i;
    float lambda2 = 1.0;

    substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_x, p, xi, lambda2, N);
    gpuErrchk(hipDeviceSynchronize());

    projection<<<numBlocksNN, threadsPerBlockNN>>>(device_xt, device_x, MINPIX, N);
    gpuErrchk(hipDeviceSynchronize());

    ft = (*func)(p);

    substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_pl, p, xi, 1.0, N);
    gpuErrchk(hipDeviceSynchronize());

    normVectorCalculation<<<numBlocksNN, threadsPerBlockNN>>>(device_normVector, device_pl, N);
    gpuErrchk(hipDeviceSynchronize());

    normPL = deviceReduce<float>(device_normVector, M*N);

    fgoal = fc * normPL *(ALPHA/lambda2);
    iarm = 0;
    while(ft<fgoal){
      lambda2 = lambda2 * 0.1;
      substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_x, p, xi, lambda2, N);
      gpuErrchk(hipDeviceSynchronize());

      projection<<<numBlocksNN, threadsPerBlockNN>>>(device_xt, device_x, MINPIX, N);
      gpuErrchk(hipDeviceSynchronize());

      substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_pl, p, device_xt, lambda2, N);
      gpuErrchk(hipDeviceSynchronize());

      ft = (*func)(p);

      normVectorCalculation<<<numBlocksNN, threadsPerBlockNN>>>(device_normVector, device_pl, N);
      gpuErrchk(hipDeviceSynchronize());

      normPL = deviceReduce<float>(device_normVector, M*N);
      iarm++;
      if(iarm>10){
        break;
      }
      fgoal = fc * normPL * (ALPHA/lambda2);
    }

    //p.x = xt
    copyImage<<<numBlocksNN, threadsPerBlockNN>>>(p, device_xt, N);
    gpuErrchk(hipDeviceSynchronize());

    fc = (*func)(p);
    printf("Function value = %f\n", fc);
    //exit(0);
    (*dfunc)(p,xi);

    substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_x, p, xi, 1.0, N);
    gpuErrchk(hipDeviceSynchronize());

    projection<<<numBlocksNN, threadsPerBlockNN>>>(device_pgc, device_x, MINPIX, N);
    gpuErrchk(hipDeviceSynchronize());

    normVectorCalculation<<<numBlocksNN, threadsPerBlockNN>>>(device_normVector, device_pgc, N);
    gpuErrchk(hipDeviceSynchronize());

    normPGC = deviceReduce<float>(device_normVector, M*N);
    i++;
    end = omp_get_wtime();
    double wall_time = end-start;
    printf("Time: %lf seconds\n", i, wall_time);
  }

  hipFree(device_normVector);
  hipFree(device_pgc);
  hipFree(device_x);
  hipFree(xi);
  hipFree(device_xt);
  hipFree(device_pl);

}


__host__ void frprmn(hipfftComplex *p, float ftol, float *fret, float (*func)(hipfftComplex*), void (*dfunc)(hipfftComplex*, float*))
{
  float gg, dgg, gam, fp;
  float *device_g, *device_h, *xi;
  double start, end;


  //////////////////////MEMORY GPU//////////////////////////
  gpuErrchk(hipMalloc((void**)&device_g, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_g, 0, sizeof(float)*M*N));
  gpuErrchk(hipMalloc((void**)&device_h, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_h, 0, sizeof(float)*M*N));
  gpuErrchk(hipMalloc((void**)&xi, sizeof(float)*M*N));
  gpuErrchk(hipMemset(xi, 0, sizeof(float)*M*N));

  ///////////////////vectors for gg and dgg////////////////////
  float *device_gg_vector, *device_dgg_vector;

  //////////////////////////////////GPU MEMORY///////////////
  gpuErrchk(hipMalloc((void**)&device_gg_vector, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_gg_vector, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_dgg_vector, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_dgg_vector, 0, sizeof(float)*M*N));

  fp = (*func)(p);
  if(verbose_flag){
    printf("Starting function value = %f\n", fp);
  }
  (*dfunc)(p,xi);
  //g=-xi
  //xi=h=g
  searchDirection<<<numBlocksNN, threadsPerBlockNN>>>(device_g, xi, device_h, N);//Search direction
  gpuErrchk(hipDeviceSynchronize());

  ////////////////////////////////////////////////////////////////
  for(int i=1; i <= it_maximum; i++){
    start = omp_get_wtime();
    iter = i;
    if(verbose_flag){
      printf("\n\n********** Iteration %d **********\n\n", i);
    }
    linmin(p, xi, fret, func);

    if (2.0*fabs(*fret-fp) <= ftol*(fabs(*fret)+fabs(fp)+EPS)) {
      printf("Exit due to tolerance\n");
      FREEALL
			return;
		}

    fp=(*func)(p);
    if(verbose_flag){
      printf("Function value = %f\n", fp);
    }
    (*dfunc)(p,xi);
    dgg = gg = 0.0;
    ////gg = g*g
    ////dgg = (xi+g)*xi
    getGandDGG<<<numBlocksNN, threadsPerBlockNN>>>(device_gg_vector, device_dgg_vector, xi, device_g, N);
  	gpuErrchk(hipDeviceSynchronize());
    ////getSums (Reductions) of gg dgg
    gg = deviceReduce<float>(device_gg_vector, M*N);
    dgg = deviceReduce<float>(device_dgg_vector, M*N);
    if(gg == 0.0){
      printf("Exit due to gg = 0\n");
      FREEALL
      return;
    }
    gam = fmax(0.0f, dgg/gg);
    //printf("Gamma = %f\n", gam);
    //g=-xi
    //xi=h=g+gam*h;
    newXi<<<numBlocksNN, threadsPerBlockNN>>>(device_g, xi, device_h, gam, N);
  	gpuErrchk(hipDeviceSynchronize());
    end = omp_get_wtime();
    double wall_time = end-start;
    if(verbose_flag){
      printf("Time: %lf seconds\n", i, wall_time);
    }
  }
  printf("Too many iterations in frprmn\n");
  FREEALL
  return;
}

__host__ void LBFGS_recursion(float *d_y, hipfftComplex* d_s, float *xi, int par_M, int lbfgs_it, int M, int N){
  float *alpha, *aux_vector;
  float *d_r, *d_q;
  float rho = 0.0f;
  float rho_den;
  float beta = 0.0f;
  float sy = 0.0f;
  float yy = 0.0f;
  float sy_yy = 0.0f;
  alpha = (float*)malloc(par_M*sizeof(float));
  memset (alpha,0,par_M*sizeof(float));

  gpuErrchk(hipMalloc((void**)&aux_vector, sizeof(float)*M*N));
  gpuErrchk(hipMalloc((void**)&d_q, sizeof(float)*M*N));
  gpuErrchk(hipMalloc((void**)&d_r, sizeof(float)*M*N));

  gpuErrchk(hipMemset(aux_vector, 0, sizeof(float)*M*N));
  gpuErrchk(hipMemset(d_r, 0, sizeof(float)*M*N));
  gpuErrchk(hipMemcpy(d_q, xi, sizeof(float)*M*N, hipMemcpyDeviceToDevice));

  for(int k=0; k<par_M; k++){
    //Rho_k = 1.0/(y_k's_k);
    getDot_LBFGS_fComplex<<<numBlocksNN, threadsPerBlockNN>>>(aux_vector, d_s, d_y, k, k, M, N);
    gpuErrchk(hipDeviceSynchronize());
    rho_den = deviceReduce<float>(aux_vector, M*N);
    if(rho_den != 0.0f)
      rho = 1.0/rho_den;
    else
      rho = 0.0f;

    //alpha_k = Rho_k x (s_k' * q);
    getDot_LBFGS_fComplex<<<numBlocksNN, threadsPerBlockNN>>>(aux_vector, d_s, d_q, k, 0, M, N);
    gpuErrchk(hipDeviceSynchronize());
    alpha[k] = rho * deviceReduce<float>(aux_vector, M*N);
    //q = q - alpha_k * y_k;
    updateQ<<<numBlocksNN, threadsPerBlockNN>>>(d_q, -alpha[k], d_y, k, M, N);
    gpuErrchk(hipDeviceSynchronize());

  }

  //s0'y_0
  getDot_LBFGS_fComplex<<<numBlocksNN, threadsPerBlockNN>>>(aux_vector, d_s, d_y, lbfgs_it, lbfgs_it, M, N);
  gpuErrchk(hipDeviceSynchronize());
  sy = deviceReduce<float>(aux_vector, M*N);
  //y_0'y_0
  getDot_LBFGS_ff<<<numBlocksNN, threadsPerBlockNN>>>(aux_vector, d_y, d_y, lbfgs_it, lbfgs_it, M, N);
  gpuErrchk(hipDeviceSynchronize());
  yy = deviceReduce<float>(aux_vector, M*N);
  //(s_0'y_0)/(y_0'y_0)
  if(yy!=0.0)
    sy_yy = sy/yy;
  else
    sy_yy = 0.0f;

  // r = q x ((s_0'y_0)/(y_0'y_0));
  getR<<<numBlocksNN, threadsPerBlockNN>>>(d_q, sy_yy, N);
  gpuErrchk(hipDeviceSynchronize());

  for (int k = par_M - 1; k >= 0; k--)
  {
    //Rho_k = 1.0/(y_k's_k);
    getDot_LBFGS_fComplex<<<numBlocksNN, threadsPerBlockNN>>>(aux_vector, d_s, d_y, k, k, M, N);
    gpuErrchk(hipDeviceSynchronize());
    //Calculate rho backwards
    rho_den = deviceReduce<float>(aux_vector, M*N);
    if(rho_den != 0.0f)
      rho = 1.0/rho_den;
    else
      rho = 0.0f;
    //beta = rho * y_k' * r;
    getDot_LBFGS_ff<<<numBlocksNN, threadsPerBlockNN>>>(aux_vector, d_y, d_r, k, 0, M, N);
    gpuErrchk(hipDeviceSynchronize());
    //beta = rho * y_k' * r;
    beta = rho * deviceReduce<float>(aux_vector, M*N);
    //r = r + s_k * (alpha_k - beta)
    updateQComplex<<<numBlocksNN, threadsPerBlockNN>>>(d_r, alpha[k]-beta, d_s, k, M, N);
  }

  searchDirection_LBFGS<<<numBlocksNN, threadsPerBlockNN>>>(d_r, N);//Search direction
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(xi, d_r, sizeof(float)*M*N, hipMemcpyDeviceToDevice));
  hipFree(aux_vector);
  hipFree(d_q);
  hipFree(d_r);
  hipFree(aux_vector);
  free(alpha);
}

__host__ void LBFGS(hipfftComplex *p, float ftol, float *fret, float (*func)(hipfftComplex*), void (*dfunc)(hipfftComplex*, float*), int K)
{
  hipfftComplex *d_s;
  float *d_y, *xi, *xi_old, *norm_vector;
  float norm, fp;
  hipfftComplex *p_old;
  double start, end;

  gpuErrchk(hipMalloc((void**)&d_y, sizeof(float)*M*N*K));
  gpuErrchk(hipMemset(d_y, 0, sizeof(float)*M*N*K));

  gpuErrchk(hipMalloc((void**)&d_s, sizeof(hipfftComplex)*M*N*K));
  gpuErrchk(hipMemset(d_s, 0, sizeof(hipfftComplex)*M*N*K));

  gpuErrchk(hipMalloc((void**)&p_old, sizeof(hipfftComplex)*M*N));
  gpuErrchk(hipMemset(p_old, 0, sizeof(hipfftComplex)*M*N));

  gpuErrchk(hipMalloc((void**)&xi, sizeof(float)*M*N));
  gpuErrchk(hipMemset(xi, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&xi_old, sizeof(float)*M*N));
  gpuErrchk(hipMemset(xi_old, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&norm_vector, sizeof(float)*M*N));
  gpuErrchk(hipMemset(norm_vector, 0, sizeof(float)*M*N));

  fp = (*func)(p);
  if(verbose_flag){
    printf("Starting function value = %f\n", fp);
  }
  (*dfunc)(p,xi);

  searchDirection_LBFGS<<<numBlocksNN, threadsPerBlockNN>>>(xi, N);//Search direction
  gpuErrchk(hipDeviceSynchronize());

  for(int i=1; i <= it_maximum; i++){
    start = omp_get_wtime();
    iter = i;
    if(verbose_flag){
      printf("\n\n********** Iteration %d **********\n\n", i);
    }

    gpuErrchk(hipMemcpy2D(p_old, sizeof(hipfftComplex), p, sizeof(hipfftComplex), sizeof(hipfftComplex), M*N, hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemcpy2D(xi_old, sizeof(float), xi, sizeof(float), sizeof(float), M*N, hipMemcpyDeviceToDevice));

    linmin(p, xi, fret, func);

    if (2.0*fabs(*fret-fp) <= ftol*(fabs(*fret)+fabs(fp)+EPS)) {
      printf("Exit due to tolerance\n");
      FREEALL_LBFGS
			return;
		}

    getDot_LBFGS_ff<<<numBlocksNN, threadsPerBlockNN>>>(norm_vector, xi, xi, 0, 0, M, N);
    gpuErrchk(hipDeviceSynchronize());
    norm = deviceReduce<float>(norm_vector, M*N);

    if(norm <= ftol){
      printf("Exit due to norm = 0\n");
      FREEALL_LBFGS
      return;
    }

    fp=(*func)(p);
    if(verbose_flag){
      printf("Function value = %f\n", fp);
    }
    (*dfunc)(p,xi);

    calculateSandY<<<numBlocksNN, threadsPerBlockNN>>>(d_s, d_y, p, xi, p_old, xi_old, (iter-1)%K, M, N);
    gpuErrchk(hipDeviceSynchronize());

    LBFGS_recursion(d_y, d_s, xi, std::min(K,iter), (iter-1)%K, M, N);

    end = omp_get_wtime();
    double wall_time = end-start;
    if(verbose_flag){
      printf("Time: %lf seconds\n", i, wall_time);
    }

  }
  printf("Too many iterations in LBFGS\n");
  FREEALL_LBFGS
  return;

}
