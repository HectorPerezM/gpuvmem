#include "hip/hip_runtime.h"
#include "frprmn.cuh"

extern long M;
extern long N;
extern int iter;

extern dim3 threadsPerBlockNN;
extern dim3 numBlocksNN;

extern int threadsVectorNN;
extern int blocksVectorNN;

extern float MINPIX;

#define EPS 1.0e-10
#define ITERATIONS 500

#define ARMIJOTOLERANCE 1.e-6
#define ALPHA 1.e-4


#define FREEALL hipFree(device_gg_vector);hipFree(device_dgg_vector);hipFree(xi);hipFree(device_h);hipFree(device_g);



__host__ void armijoTest(hipfftComplex *p, float (*func)(hipfftComplex*), void (*dfunc)(hipfftComplex*, float*))
{
  int i = 0;
  int iarm;
  float normPGC, normPL, fc, ft, fgoal;
  float *device_normVector, *device_pgc, *device_x, *xi, *device_xt, *device_pl;

  gpuErrchk(hipMalloc((void**)&device_normVector, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_normVector, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_pgc, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_pgc, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_x, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_x, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&xi, sizeof(float)*M*N));
  gpuErrchk(hipMemset(xi, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_xt, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_xt, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_pl, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_pl, 0, sizeof(float)*M*N));


  fc = (*func)(p);
  printf("Function value = %f\n", fc);
  //exit(0);
  (*dfunc)(p,xi);

  substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_x, p, xi, 1.0, N);
  gpuErrchk(hipDeviceSynchronize());

  projection<<<numBlocksNN, threadsPerBlockNN>>>(device_xt, device_x, MINPIX, N);
  gpuErrchk(hipDeviceSynchronize());

  substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_pgc, p, device_xt, 1.0, N);
  gpuErrchk(hipDeviceSynchronize());

  normVectorCalculation<<<numBlocksNN, threadsPerBlockNN>>>(device_normVector, device_pgc, N);
  gpuErrchk(hipDeviceSynchronize());

  normPGC = deviceReduce(device_normVector, M*N);
  i=1;
  while(normPGC > ARMIJOTOLERANCE && i <= ITERATIONS){
    iter = i;
    float lambda2 = 1.0;

    substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_x, p, xi, lambda2, N);
    gpuErrchk(hipDeviceSynchronize());

    projection<<<numBlocksNN, threadsPerBlockNN>>>(device_xt, device_x, MINPIX, N);
    gpuErrchk(hipDeviceSynchronize());

    ft = (*func)(p);

    substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_pl, p, xi, 1.0, N);
    gpuErrchk(hipDeviceSynchronize());

    normVectorCalculation<<<numBlocksNN, threadsPerBlockNN>>>(device_normVector, device_pl, N);
    gpuErrchk(hipDeviceSynchronize());

    normPL = deviceReduce(device_normVector, M*N);

    fgoal = fc * normPL *(ALPHA/lambda2);
    iarm = 0;
    while(ft<fgoal){
      lambda2 = lambda2 * 0.1;
      substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_x, p, xi, lambda2, N);
      gpuErrchk(hipDeviceSynchronize());

      projection<<<numBlocksNN, threadsPerBlockNN>>>(device_xt, device_x, MINPIX, N);
      gpuErrchk(hipDeviceSynchronize());

      substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_pl, p, device_xt, lambda2, N);
      gpuErrchk(hipDeviceSynchronize());

      ft = (*func)(p);

      normVectorCalculation<<<numBlocksNN, threadsPerBlockNN>>>(device_normVector, device_pl, N);
      gpuErrchk(hipDeviceSynchronize());

      normPL = deviceReduce(device_normVector, M*N);
      iarm++;
      if(iarm>10){
        break;
      }
      fgoal = fc * normPL * (ALPHA/lambda2);
    }

    //p.x = xt
    copyImage<<<numBlocksNN, threadsPerBlockNN>>>(p, device_xt, N);
    gpuErrchk(hipDeviceSynchronize());

    fc = (*func)(p);
    printf("Function value = %f\n", fc);
    //exit(0);
    (*dfunc)(p,xi);

    substraction<<<numBlocksNN, threadsPerBlockNN>>>(device_x, p, xi, 1.0, N);
    gpuErrchk(hipDeviceSynchronize());

    projection<<<numBlocksNN, threadsPerBlockNN>>>(device_pgc, device_x, MINPIX, N);
    gpuErrchk(hipDeviceSynchronize());

    normVectorCalculation<<<numBlocksNN, threadsPerBlockNN>>>(device_normVector, device_pgc, N);
    gpuErrchk(hipDeviceSynchronize());

    normPGC = deviceReduce(device_normVector, M*N);
    i++;
  }

  hipFree(device_normVector);
  hipFree(device_pgc);
  hipFree(device_x);
  hipFree(xi);
  hipFree(device_xt);
  hipFree(device_pl);

}


__host__ void frprmn(hipfftComplex *p, float ftol, float *fret, float (*func)(hipfftComplex*), void (*dfunc)(hipfftComplex*, float*))
{
  float gg, dgg, gam, fp;
  float *device_g, *device_h, *xi;


  //////////////////////MEMORY GPU//////////////////////////
  gpuErrchk(hipMalloc((void**)&device_g, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_g, 0, sizeof(float)*M*N));
  gpuErrchk(hipMalloc((void**)&device_h, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_h, 0, sizeof(float)*M*N));
  gpuErrchk(hipMalloc((void**)&xi, sizeof(float)*M*N));
  gpuErrchk(hipMemset(xi, 0, sizeof(float)*M*N));

  ///////////////////vectors for gg and dgg////////////////////
  float *device_gg_vector, *device_dgg_vector;

  //////////////////////////////////GPU MEMORY///////////////
  gpuErrchk(hipMalloc((void**)&device_gg_vector, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_gg_vector, 0, sizeof(float)*M*N));

  gpuErrchk(hipMalloc((void**)&device_dgg_vector, sizeof(float)*M*N));
  gpuErrchk(hipMemset(device_dgg_vector, 0, sizeof(float)*M*N));

  fp = (*func)(p);
  printf("Function value = %f\n", fp);
  (*dfunc)(p,xi);
  //g=-xi
  //xi=h=g
  searchDirection<<<numBlocksNN, threadsPerBlockNN>>>(device_g, xi, device_h, N);//Search direction
  gpuErrchk(hipDeviceSynchronize());



  ////////////////////////////////////////////////////////////////
  for(int i=1; i <= ITERATIONS; i++){
    iter = i;
    printf("\n\n**********Iteration %d **********\n\n", i);
    linmin(p, xi, fret, func);

    if (2.0*fabs(*fret-fp) <= ftol*(fabs(*fret)+fabs(fp)+EPS)) {
      printf("Exit due to tolerance\n");
      FREEALL
			return;
		}

    fp=(*func)(p);
    printf("Function value = %f\n", fp);
    (*dfunc)(p,xi);
    dgg = gg = 0.0;
    ////gg = g*g
    ////dgg = (xi+g)*xi
    getGandDGG<<<numBlocksNN, threadsPerBlockNN>>>(device_gg_vector, device_dgg_vector, xi, device_g, N);
  	gpuErrchk(hipDeviceSynchronize());
    ////getSums (Reductions) of gg dgg
    gg = deviceReduce(device_gg_vector, M*N);
    dgg = deviceReduce(device_dgg_vector, M*N);
    if(gg == 0.0){
      printf("Exit due to gg = 0\n");
      FREEALL
      return;
    }
    gam = dgg/gg;
    //printf("Gamma = %f\n", gam);
    //g=-xi
    //xi=h=g+gam*h;
    newXi<<<numBlocksNN, threadsPerBlockNN>>>(device_g, xi, device_h, gam, N);
  	gpuErrchk(hipDeviceSynchronize());

  }
  printf("Too many iterations in frprmn\n");
  FREEALL
  return;
}
