#include "f1dim.cuh"
extern hipfftComplex *device_pcom;
extern float *device_xicom, (*nrfunc)(hipfftComplex*);
extern long M;
extern long N;
extern float MINPIX;
extern dim3 threadsPerBlockNN;
extern dim3 numBlocksNN;
extern int nopositivity;

__host__ float f1dim(float x)
{
    hipfftComplex *device_xt;
    float f;

    gpuErrchk(hipMalloc((void**)&device_xt, sizeof(hipfftComplex)*M*N));
    gpuErrchk(hipMemset(device_xt, 0, sizeof(hipfftComplex)*M*N));

    //printf("Se evalua en f1dim %f\n", x);
    //xt = pcom+x*xicom;
    if(nopositivity == 0){
      evaluateXt<<<numBlocksNN, threadsPerBlockNN>>>(device_xt, device_pcom, device_xicom, x, MINPIX, N);
      gpuErrchk(hipDeviceSynchronize());
    }else{
      evaluateXtNoPositivity<<<numBlocksNN, threadsPerBlockNN>>>(device_xt, device_pcom, device_xicom, x, N);
      gpuErrchk(hipDeviceSynchronize());
    }

    f = (*nrfunc)(device_xt);
    hipFree(device_xt);
    return f;
}
