#include "squaredtotalvariation.cuh"

extern long M, N;
extern int image_count;
extern float * penalizators;
extern int nPenalizators;

SquaredTotalVariationP::SquaredTotalVariationP(){
};

float SquaredTotalVariationP::calcFi(float *p)
{
        float result = 0.0;
        result = (penalization_factor)*(squaredTotalVariation(p, device_S, penalization_factor, mod, order, imageIndex) );
        return result;
}
void SquaredTotalVariationP::calcGi(float *p, float *xi)
{
        DSTVariation(p, device_DS, penalization_factor, mod, order, imageIndex);
};


void SquaredTotalVariationP::restartDGi()
{
        gpuErrchk(hipMemset(device_DS, 0, sizeof(float)*M*N));
};

void SquaredTotalVariationP::addToDphi(float *device_dphi)
{
        linkAddToDPhi(device_dphi, device_DS, imageToAdd);
};

void SquaredTotalVariationP::configure(int penalizatorIndex, int imageIndex, int imageToAdd)
{
        this->imageIndex = imageIndex;
        this->order = order;
        this->mod = mod;
        this->imageToAdd = imageToAdd;

        if(imageIndex > image_count -1 || imageToAdd > image_count -1)
        {
                printf("There is no image for the provided index (SquaredTotalVariationP)\n");
                exit(-1);
        }

        if(penalizatorIndex != -1)
        {
                if(penalizatorIndex > (nPenalizators - 1) || penalizatorIndex < 0)
                {
                        printf("invalid index for penalizator (SquaredTotalVariationP)\n");
                        exit(-1);
                }else{
                        this->penalization_factor = penalizators[penalizatorIndex];
                }
        }

        gpuErrchk(hipMalloc((void**)&device_S, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_S, 0, sizeof(float)*M*N));

        gpuErrchk(hipMalloc((void**)&device_DS, sizeof(float)*M*N));
        gpuErrchk(hipMemset(device_DS, 0, sizeof(float)*M*N));

};

void SquaredTotalVariationP::setSandDs(float *S, float *Ds)
{
        hipFree(this->device_S);
        hipFree(this->device_DS);
        this->device_S = S;
        this->device_DS = Ds;
};

namespace {
Fi* CreateSquaredTotalVariationP()
{
        return new SquaredTotalVariationP;
}
const int SquaredTotalVariationPId = 5;
const bool RegisteredSquaredTotalVariationP = Singleton<FiFactory>::Instance().RegisterFi(SquaredTotalVariationPId, CreateSquaredTotalVariationP);
};
